#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include <thrust/device_vector.h>


#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"
#include <stream_compaction/stream_compaction.h>


#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__ thrust::default_random_engine makeSeededRandomEngine(int iter, int index = 0, int depth = 0) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene *hst_scene = NULL;
static glm::vec3 *dev_image = NULL;
// TODO: static variables for device memory, scene/camera info, etc
// ...

static Path * dev_path;

//static Ray * dev_ray0;
//static Ray * dev_ray1;

//static thrust::device_vector<Ray> * dev_ray0;

//static Ray * dev_ray_cur;
//static Ray * dev_ray_next;

//static thrust::device_vector<Geom> dev_geom;			//global memory
//static thrust::device_vector<Material> dev_material;	//global
static Geom * dev_geom;
static Material * dev_material;



//kd tree structure
static Node * dev_node;		//kd tree node

static int * dev_geom_idx;



void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));
    // TODO: initialize the above static variables added above

	

	hipMalloc(&dev_path, pixelcount * sizeof(Path));

	

	hipMalloc(&dev_geom, scene->geoms.size() * sizeof (Geom));
	hipMemcpy(dev_geom, scene->geoms.data() , scene->geoms.size() * sizeof (Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_material,scene->geoms.size() * sizeof(Material));
	hipMemcpy(dev_material,scene->materials.data() , scene->materials.size() * sizeof (Material), hipMemcpyHostToDevice);



	//KD-tree
	

	hipMalloc(&dev_node, (scene->kdtree.hst_node.size()) * sizeof(Node) );
	hipMemcpy(dev_node, scene->kdtree.hst_node.data(), (scene->kdtree.hst_node.size()) * sizeof(Node), hipMemcpyHostToDevice);

	hipMalloc(&dev_geom_idx, (scene->kdtree.hst_geom_idx.size()) * sizeof(int));
	hipMemcpy(dev_geom_idx, scene->kdtree.hst_geom_idx.data(), (scene->kdtree.hst_geom_idx.size()) * sizeof(int), hipMemcpyHostToDevice);


    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
    // TODO: clean up the above static variables

	hipFree(dev_node);
	hipFree(dev_geom_idx);
	
	hipFree(dev_path);

	hipFree(dev_geom);
	hipFree(dev_material);

    checkCUDAError("pathtraceFree");
}


__host__ __device__ void getCameraRayAtPixel(Path & path,const Camera &c, int x, int y,int iter,int index)
{
	thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
	thrust::uniform_real_distribution<float> u01(0, 1);


	path.ray.origin = c.position;
	path.ray.direction = glm::normalize(c.view
		- c.right * c.pixelLength.x * ((float)x - (float)c.resolution.x * 0.5f + u01(rng))  		//u01(rng) is for jiitering for antialiasing
		- c.up * c.pixelLength.y * ((float)y - (float)c.resolution.y * 0.5f + u01(rng)) 			//u01(rng) is for jiitering for antialiasing
		);

	if (c.lensRadiaus > 0)
	{
		//lens effect
		float r = c.lensRadiaus * u01(rng);
		float theta = u01(rng) * 2 * PI;

		
		float t = c.focalDistance * c.view.z / path.ray.direction.z;

		glm::vec3 pfocus = path.ray.origin + t * path.ray.direction;

		path.ray.origin = c.position + c.right * r * cos(theta) - c.up * r * sin(theta);
		path.ray.direction = glm::normalize(pfocus - path.ray.origin);
	}
	
	path.image_index = index;
	path.color = glm::vec3(1.0f);
	path.terminated = false;
	
}


/**
 * Generate Rays from camera through screen to the field
 * which is the first generation of rays
 *
 * Antialiasing - num of rays per pixel
 * motion blur - jitter scene position
 * lens effect - jitter camera position
 */
__global__ void generateRayFromCamera(Camera cam, int iter, Path* paths)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
		Path & path = paths[index];
		getCameraRayAtPixel(path,cam,x,y,iter,index);


    }
}



//__device__ void hitTestGeomsNaive(int geoms_size,Path & path,Geom * geoms,glm::vec3 & intersect_point, glm::vec3 & normal,int & hit_geom_index)
//{
//	float t;
//	float t_min = FLT_MAX;
//	for(int i = 0; i < geoms_size; i++)
//	{
//		//Geom & geom = static_cast<Geom>(*it);
//		glm::vec3 tmp_intersect;
//		glm::vec3 tmp_normal;
//		Geom geom = geoms[i];
//		if( geom.type == CUBE)
//		{
//			t = boxIntersectionTest(geom,path.ray,tmp_intersect,tmp_normal);
//		}
//		else if( geom.type == SPHERE)
//		{
//			t = sphereIntersectionTest(geom,path.ray,tmp_intersect,tmp_normal);
//		}
//		else
//		{
//			//TODO: triangle
//			printf("ERROR: geom type error at %d\n",i);
//		}
//
//		if(t > 0 && t_min > t)
//		{
//			t_min = t;
//			hit_geom_index = i;
//			intersect_point = tmp_intersect;
//			normal = tmp_normal;
//		}
//	}
//}


__device__ int kd_search_leaf(int & cur_idx, Node * nodes, Geom* geoms, int * geomsid
	                          ,const Ray & ray,float& tmin, float& tmax, float global_tmax
							  ,glm::vec3 & intersect, glm::vec3 & normal, bool & outside)
{
	//search for a hit in this leaf
	Node & n = nodes[cur_idx];
	float t;
	glm::vec3 leaf_intersect_point;
	glm::vec3 leaf_normal;
	float t_min = FLT_MAX;
	int hit_geom_index = -1;
	bool leaf_outside = true;

	for (int i = 0; i < n.num_geoms; i++)
	{
		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;
		bool tmp_outside = true;
		int gid = geomsid[n.geom_index] + i;
		Geom & geom = geoms[gid];
		if (geom.type == CUBE)
		{
			t = boxIntersectionTest(geom, ray, tmp_intersect, tmp_normal, tmp_outside);
		}
		else if (geom.type == SPHERE)
		{
			t = sphereIntersectionTest(geom, ray, tmp_intersect, tmp_normal, tmp_outside);
		}
		else
		{
			// triangle
			t = triangleIntersectionTest(geom, ray, tmp_intersect, tmp_normal, tmp_outside);
		}

		if (t > 0 && t_min > t)
		{
			t_min = t;
			hit_geom_index = gid;
			leaf_intersect_point = tmp_intersect;
			leaf_normal = tmp_normal;
			leaf_outside = tmp_outside;
		}
	}



	//////////////////////////////////////////////////////////////////////
	if(t > 0 && t < tmax )
	{
		// found hithit
		intersect = leaf_intersect_point;
		normal = leaf_normal;
		outside = leaf_outside;
		return hit_geom_index;
	}
	else
	{
		//continue search
		if (fabs(tmax - global_tmax) < RAY_EPSILON)
		{
			//fail, no collision
			//end search
			return -1;
		}
		else
		{
			float tmp_tmin = tmax,tmp_tmax = global_tmax;
			float t0, t1;

			//backtrack
			bool tmp_hit = AABBIntersect(n.aabb,ray,t0,t1);
			int backtrack_idx = cur_idx;
			if (!(t0 >= tmp_tmin && t1 <= tmp_tmax))
			{
				tmp_hit = false;
			}

			while(!tmp_hit)
			{
				//tmp_tmin = tmax;
				//tmp_tmax = global_tmax;
				
				//call backtrack again
				backtrack_idx = nodes[backtrack_idx].parent_idx;

				if (backtrack_idx < 0)
				{
					//error...
					//should happen
					printf("ERROR: kd tree backtrack to root!\n");
					return -1;
				}

				tmp_hit = AABBIntersect(nodes[backtrack_idx].aabb,ray,t0,t1);
				if (! (t0 >= tmp_tmin && t1 <= tmp_tmax ) )
				{
					tmp_hit = false;
				}
			}
			
			//has intersection
			cur_idx = backtrack_idx;
			tmin = t0;
			tmax = t1;
			
			return -2;
		}
	}
}


__device__ int kd_search_split(int & cur_idx,Node & n,const Ray & ray,float& tmin, float& tmax)
{
	float thit = (n.split.pos - ray.origin[n.split.axis]) / ray.direction[n.split.axis];
	int first,second;
	//order
	if(ray.direction[n.split.axis] > 0.0f)
	{
		//first = n.left_idx;
		first = cur_idx + 1;
		second = n.right_idx;
	}
	else
	{
		first = n.right_idx;
		//second = n.left_idx;
		second = cur_idx + 1;
	}


	if(thit >= tmax || thit < 0)
	{
		cur_idx = first;
	}
	else if( thit <= tmin)
	{
		cur_idx = second;
	}
	else
	{
		cur_idx = first;
		tmax = thit;
	}


	return -2;
}


//return:
//-1 end, no collision
//-2 continue
//>=0 hit_geom_id
__device__ int kd_serach_node(int & cur_idx,Node * nodes,Geom* geoms, int * geomsid
							  ,const Ray & ray,float& tmin,float& tmax, float global_tmax
							  ,glm::vec3 & intersect, glm::vec3 & normal, bool & outside)
{
	if(nodes[cur_idx].geom_index == -1)
	{
		//internal node
		return kd_search_split(cur_idx,nodes[cur_idx],ray, tmin, tmax);
		
	}
	else
	{
		//leaf node
		return kd_search_leaf(cur_idx, nodes, geoms, geomsid
			, ray, tmin, tmax, global_tmax
			, intersect, normal, outside);
	}
}

//__device__ void kd_search_init(int root_idx,Node * nodes,const Ray & ray)
//{
//	float tmin,tmax;
//	AABBIntersect(nodes[root_idx].aabb,ray,tmin,tmax);
//	kd_serach_node(root_idx,nodes,ray,tmin,tmax);
//}




__global__ void pathTraceOneBounce(int iter, int depth,int num_paths,glm::vec3 * image
										,Path * paths
										,Geom * geoms, int geoms_size
										,Material * materials, int materials_size
										,Node * nodes
										, int * geomsid
										//,const thrust::device_vector<Geom> & geoms , const thrust::device_vector<Material> & materials
										)
{
	//int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	//int path_index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	int path_index =  blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	
	if(path_index < num_paths)
	{
		Path & path = paths[path_index];	//TODO: reconsider the speed for the memory access here
		//Path & path = paths[path_index];
		//calculate intersection
		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;


#ifndef USE_KDTREE_FLAG
		//naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			//Geom & geom = static_cast<Geom>(*it);
			glm::vec3 tmp_intersect;
			glm::vec3 tmp_normal;
			Geom & geom = geoms[i];
			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, path.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, path.ray, tmp_intersect, tmp_normal, outside);
			}
			else
			{
				// triangle
				//printf("ERROR: geom type error at %d\n",i);
				t = triangleIntersectionTest(geom, path.ray, tmp_intersect, tmp_normal, outside);
			}

			if (t > 0 && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}
		

		///////////////////////////////
#else

		//TODO:k-d tree traverse approach

		int state = -2;
		int cur_idx = 0;		//tmp, root node always 0....
		float global_tmin, global_tmax;
		AABBIntersect(nodes[cur_idx].aabb, path.ray, global_tmin, global_tmax);
		float tmin = global_tmin, tmax = global_tmax;
		while (state == -2)
		{
			AABBIntersect(nodes[cur_idx].aabb, path.ray, tmin, tmax);
			state = kd_serach_node(cur_idx, nodes, geoms, geomsid
				,path.ray, tmin, tmax, global_tmax
				, intersect_point, normal, outside);

		}
		hit_geom_index = state;

		////////////////////////////
#endif


		if (hit_geom_index == -1)
		{
			path.terminated = true;
			image[path.image_index] += BACKGROUND_COLOR;
		}
		else
		{
			//hit something
			Geom & geom = geoms[hit_geom_index];
			Material & material = materials[geom.materialid];


			//if (geom.type == TRIANGLE)
			//{
			//	path.terminated = true;
			//	image[path.image_index] += glm::vec3(1.0f);
			//	return;
			//}


			if (material.emittance > EPSILON)
			{
				//light source
				path.terminated = true;
				image[path.image_index] += path.color * material.color * material.emittance;
			}
			else
			{
				path.terminated = false;
				thrust::default_random_engine rng = makeSeededRandomEngine(iter, path.image_index, depth);
				scatterRay(path.ray, path.color, intersect_point, normal, material, rng);
			}



		}


		
	}
}


struct is_path_terminated
{
  __host__ __device__
  bool operator()(const Path path)
  {
	  return path.terminated;
  }
};

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    const int blockSideLength = 8;
    const dim3 blockSize(blockSideLength, blockSideLength);
	const int blockSizeTotal = blockSideLength * blockSideLength;
    
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray is a (ray, color) pair, where color starts as the
    //     multiplicative identity, white = (1, 1, 1).
    //   * For debugging, you can output your ray directions as colors.
    // * For each depth:
    //   * Compute one new (ray, color) pair along each path (using scatterRay).
    //     Note that many rays will terminate by hitting a light or hitting
    //     nothing at all. You'll have to decide how to represent your path rays
    //     and how you'll mark terminated rays.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //       surface.
    //     * You can debug your ray-scene intersections by displaying various
    //       values as colors, e.g., the first surface normal, the first bounced
    //       ray direction, the first unlit material color, etc.
    //   * Add all of the terminated rays' results into the appropriate pixels.
    //   * Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    // * Finally, handle all of the paths that still haven't terminated.
    //   (Easy way is to make them black or background-colored.)

    // TODO: perform one iteration of path tracing


    //generateNoiseDeleteMe<<<blocksPerGrid, blockSize>>>(cam, iter, dev_image);

	int depth = 0;

	generateRayFromCamera<<<blocksPerGrid2d,blockSize>>>(cam,iter,dev_path);
	checkCUDAError("generate camera ray");

	
	Path* dev_path_end = dev_path + pixelcount;
	int num_path = dev_path_end - dev_path;
	//loop
	while (/*dev_path_end != dev_path*/num_path > 0 && depth < traceDepth)
	{
		
		dim3 blocksNeeded = (num_path + blockSizeTotal - 1) / blockSizeTotal ;
		pathTraceOneBounce<<<blocksNeeded,blockSize>>>(iter,depth, num_path  ,dev_image, dev_path
			, dev_geom, hst_scene->geoms.size()
			, dev_material, hst_scene->materials.size()
			, dev_node, dev_geom_idx);
		checkCUDAError("trace one bounce");
		hipDeviceSynchronize();
		depth ++;

		//stream compaction
		dev_path_end = thrust::remove_if(thrust::device, dev_path, dev_path_end, is_path_terminated() );
		num_path = dev_path_end - dev_path;

		//TODO:self work efficient
		//num_path = StreamCompaction::Efficient::compact(num_path, dev_path);
		
		checkCUDAError("stream compaction");
	}

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
