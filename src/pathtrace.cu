#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/device_vector.h>

#include <stream_compaction/efficient.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ANTIALIASING 0
#define DOF 0
#define USETHRUSTCOMPACTION 0
#define TIMEFUNCTION 0
#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
#define ERRORCHECK 0
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
	+ hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
	exit(EXIT_FAILURE);
#endif ERRORCHECK
}

__host__ __device__ thrust::default_random_engine random_engine(
        int iter, int index = 0, int depth = 0) {
    return thrust::default_random_engine(utilhash((index + 1) * iter) ^ utilhash(depth));
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene *hst_scene = NULL;
__constant__ static glm::vec3 *dev_image = NULL;
__constant__ static Geom* dev_geoms = NULL;
__constant__ static Material* dev_materials = NULL;
__constant__ static glm::vec3 *dev_oversample_image = NULL;
static int geomcount = 0;
static int oversampling_pass = 3;

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));
	hipMalloc(&dev_oversample_image, pixelcount * sizeof(glm::vec3));

	Geom* hst_geoms = hst_scene->geoms.data();
	Material* hst_materials = hst_scene->materials.data();

	geomcount = hst_scene->geoms.size();

	hipMalloc((void**)&dev_geoms, hst_scene->geoms.size()*sizeof(Geom));
	hipMalloc((void**)&dev_materials, hst_scene->materials.size()*sizeof(Material));
	hipMemcpy(dev_geoms, hst_geoms, hst_scene->geoms.size()*sizeof(Geom), hipMemcpyHostToDevice);
	hipMemcpy(dev_materials, hst_materials, hst_scene->materials.size()*sizeof(Material), hipMemcpyHostToDevice);

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_geoms);
	hipFree(dev_materials);
	hipFree(dev_oversample_image);

    checkCUDAError("pathtraceFree");
}

__global__ void initRayGrid(PathRay *oGrid, const Camera cam){
	// From camera as single point, to image grid with FOV
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		//PathRay pr;
		oGrid[index].index = index;
		oGrid[index].color = glm::vec3(1.0f);

		oGrid[index].ray.origin = cam.position;
		oGrid[index].terminate = false;
		oGrid[index].matId = -1;

		// Grid center to pixel
		float pX = x - cam.resolution.x / 2;
		float pY = cam.resolution.y / 2 - y;

		// Vector: grid center to pixel
		glm::vec3 o2px = cam.right*pX + cam.up*pY;
		//glm::vec3 o2px = glm::vec3(cam.right.x*pX + cam.up.x*pY, cam.right.y*pX + cam.up.y*pY, cam.right.z*pX + cam.up.z*pY);
		// Ray vector
		oGrid[index].ray.direction = cam.toGrid + o2px;

		//oGrid[index] = pr;

		// Ray direction debug
		//float l = glm::length(ray.ray.direction);
		//image[index] += glm::vec3(abs(ray.ray.direction.x / l), abs(ray.ray.direction.y / l), 0);
	}
}


__global__ void interesect(PathRay *grid, const Geom *iGeoms, const Camera cam, const int grid_size, const int geomcount){
	// From camera as single point, to image grid with FOV
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	int index = x + (y * cam.resolution.x);

	extern __shared__ Geom geoms[];

	int bIndex = threadIdx.x + threadIdx.y * blockDim.x;
	if (bIndex < geomcount){
		geoms[bIndex] = iGeoms[bIndex];
	}

	__syncthreads();

	if (index < grid_size) {
		// Intersection test
		//PathRay pr = grid[index];
		grid[index].hasIntersect = false;
		glm::vec3 iPoint(0.0f);
		glm::vec3 iNormal(0.0f);

		float rayLength = 0.0f;
		float oldLength = -1.0f;
		int idx = 0;
		bool outside = false;
		for (int i = 0; i < geomcount; ++i){
			if (geoms[i].type == SPHERE){
				rayLength = sphereIntersectionTest(geoms[i], grid[index].ray, iPoint, iNormal, outside);
			}
			else {
				rayLength = boxIntersectionTest(geoms[i], grid[index].ray, iPoint, iNormal, outside);
			}
			// Find the nearest intersection
			if (rayLength != -1.0f){
				grid[index].hasIntersect = true;
				if (oldLength == -1.0f || rayLength < oldLength){
					grid[index].intersect = iPoint;
					grid[index].normal = iNormal;
					grid[index].outside = outside;
					oldLength = rayLength;
					idx = i;
				}
			}
			/*
			Geom g = iGeoms[i];
			if (g.type == SPHERE){
				rayLength = sphereIntersectionTest(g, pr.ray, iPoint, iNormal, outside);
			} else {
				rayLength = boxIntersectionTest(g, pr.ray, iPoint, iNormal, outside);
			}
			// Find the nearest intersection
			if (rayLength != -1.0f){
				pr.hasIntersect = true;
				if (oldLength == -1.0f || rayLength < oldLength){
					oldLength = rayLength;
					pr.intersect = iPoint;
					pr.normal = iNormal;
					pr.matId = g.materialid;
					pr.outside = outside;
				}
			}
			*/
		}
		grid[index].matId = geoms[idx].materialid;
		//grid[index] = pr;
	}
};

__global__ void scatter(PathRay *grid, const Material *iMaterials, const Camera cam, const int grid_size, const int iter, const int depth){
	// From camera as single point, to image grid with FOV
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	int index = x + (y * cam.resolution.x);

	__syncthreads();

	if (index < grid_size) {
		//PathRay pr = grid[index];
		//Material m = iMaterials[pr.matId];
		scatterRay(grid[index], iMaterials[grid[index].matId], random_engine(iter, index, depth));
		//grid[index] = pr;
	}
};

__global__ void terminatePath(PathRay *grid, const Material *iMaterials, const Camera cam, const int grid_size){
	// From camera as single point, to image grid with FOV
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	int index = x + (y * cam.resolution.x);

	if (index < grid_size) {
		if (grid[index].hasIntersect){
			// Hits a light
			if (iMaterials[grid[index].matId].emittance > 0.0f){
				grid[index].terminate = true;
				grid[index].color = grid[index].color * iMaterials[grid[index].matId].color * iMaterials[grid[index].matId].emittance;
			}
		}
		else {
			// No intersections
			grid[index].terminate = true;
			grid[index].color = glm::vec3(0.0f);
		}
		//grid[index] = pr;
	}
}

__global__ void fillPixel(const PathRay *grid, glm::vec3 *image, const Camera cam, const int grid_size){
	// From camera as single point, to image grid with FOV
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	int index = x + (y * cam.resolution.x);

	if (index < grid_size) {
		PathRay pr = grid[index];
		if (pr.terminate){
			image[pr.index] += pr.color;
		}
	}
}

#if ANTIALIASING
__global__ void avgOversample(glm::vec3 *oImage, const glm::vec3 *tempImage, const Camera cam, const int passes){
	// From camera as single point, to image grid with FOV
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		oImage[index] += tempImage[index] / (float)passes;
	}
}

__global__ void jitterRay(PathRay *grid, thrust::default_random_engine rng, const Camera cam){
	// From camera as single point, to image grid with FOV
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathRay pr = grid[index];
#if DOF
		thrust::uniform_real_distribution<float> u01(-0.1, 0.1);
		// Find intersection to focal plane
		glm::vec3 p = pr.ray.origin + glm::normalize(pr.ray.direction) * cam.dof;
		glm::vec3 jitter = cam.up*u01(rng) + cam.right*u01(rng);
		// Jitter ray origin
		pr.ray.origin = pr.ray.origin + jitter;
		// Update ray direction based on new origin
		// So new ray always points to exact pixel cell on focal plane
		pr.ray.direction = p - pr.ray.origin;
#else
		thrust::uniform_real_distribution<float> u01(-0.01, 0.01);
		pr.ray.origin = glm::vec3(pr.ray.origin.x + u01(rng), pr.ray.origin.y + u01(rng), pr.ray.origin.z + u01(rng));
#endif DOF
		grid[index] = pr;
	}
}
#endif ANTIALIASING

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

	const int blockSideLength = 8;
    const dim3 blockSize(blockSideLength, blockSideLength);
    dim3 blocksPerGrid(
            (cam.resolution.x + blockSize.x - 1) / blockSize.x,
            (cam.resolution.y + blockSize.y - 1) / blockSize.y);

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray is a (ray, color) pair, where color starts as the
    //     multiplicative identity, white = (1, 1, 1).
    //   * For debugging, you can output your ray directions as colors.
    // * For each depth:
    //   * Compute one new (ray, color) pair along each path (using scatterRay).
    //     Note that many rays will terminate by hitting a light or hitting
    //     nothing at all. You'll have to decide how to represent your path rays
    //     and how you'll mark terminated rays.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //       surface.
    //     * You can debug your ray-scene intersections by displaying various
    //       values as colors, e.g., the first surface normal, the first bounced
    //       ray direction, the first unlit material color, etc.
    //   * Add all of the terminated rays' results into the appropriate pixels.
    //   * Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    // * Finally, handle all of the paths that still haven't terminated.
    //   (Easy way is to make them black or background-colored.)

    // Perform one iteration of path tracing

	thrust::device_vector<PathRay> dev_grid(pixelcount);
	PathRay *dev_grid_ptr = thrust::raw_pointer_cast(&dev_grid[0]);

#if ANTIALIASING
	hipMemset(dev_oversample_image, 0, pixelcount * sizeof(glm::vec3));
	for (int a = 0; a < oversampling_pass; a++){
		dev_grid.resize(pixelcount);
		// initRayGrid
		initRayGrid << <blocksPerGrid, blockSize >> >(dev_grid_ptr, cam);
		int grid_size = dev_grid.size();
		// Jitter for antialiasing oversampling; also accounts for DOF effect if enabled
		jitterRay << <blocksPerGrid, blockSize >> >(dev_grid_ptr, random_engine(iter, 0, oversampling_pass), cam);
#else
	// initRayGrid
	initRayGrid << <blocksPerGrid, blockSize >> >(dev_grid_ptr, cam);
	int grid_size = dev_grid.size();
#endif ANTIALIASING
	// For each traceDepth
	for (int d = 0; d < traceDepth; d++){

#if TIMEFUNCTION
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start);
#endif TIMEFUNCTION

		// Intersection test
		interesect << <blocksPerGrid, blockSize, geomcount*sizeof(Geom) >> >(dev_grid_ptr, dev_geoms, cam, grid_size, geomcount);
		checkCUDAError("intersect");

		// Mark all terminated paths
		terminatePath << <blocksPerGrid, blockSize >> >(dev_grid_ptr, dev_materials, cam, grid_size);
		checkCUDAError("terminatePath");

		// Paint image
#if ANTIALIASING
		fillPixel << <blocksPerGrid, blockSize >> >(dev_grid_ptr, dev_oversample_image, cam, grid_size);
#else
		fillPixel << <blocksPerGrid, blockSize >> >(dev_grid_ptr, dev_image, cam, grid_size);
#endif ANTIALIASING
		checkCUDAError("fillPixel");

		// Stream compaction
#if USETHRUSTCOMPACTION
		thrust::detail::normal_iterator<thrust::device_ptr<PathRay>> newGridEnd = thrust::remove_if(dev_grid.begin(), dev_grid.end(), is_terminated());
		checkCUDAError("thrustCompact");
		dev_grid.erase(newGridEnd, dev_grid.end());
		grid_size = dev_grid.size(); 
#else
		grid_size = StreamCompaction::Efficient::compact(grid_size, dev_grid_ptr);
		checkCUDAError("efficientCompact");
#endif USETHRUSTCOMPACTION

		// Scatter
		scatter << <blocksPerGrid, blockSize >> >(dev_grid_ptr, dev_materials, cam, grid_size, iter, d);
		checkCUDAError("scatter");

		// Stream compaction stat
		//printf("Iter: %d / Depth: %d / Grid size: %d\n", iter, d, grid_size);

#if TIMEFUNCTION
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		float ms1 = 0;
		hipEventElapsedTime(&ms1, start, stop);

		// Exec time comparison: thrust vs. work-efficient
		printf("Iter: %d / Depth: %d / Grid size: %d / Thrust?: %d / Time: %f\n", iter, d, grid_size, USETHRUSTCOMPACTION, ms1);
#endif TIMEFUNCTION
	}
	dev_grid.clear();
#if ANTIALIASING
	}
	// Average oversampled colors and fill into image
	avgOversample << <blocksPerGrid, blockSize >> >(dev_image, dev_oversample_image, cam, oversampling_pass);
#endif ANTIALIASING

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid, blockSize>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
