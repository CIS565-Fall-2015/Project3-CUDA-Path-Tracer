#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>

#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"

#include <stream_compaction/shared.h>

#include "sceneStructs.h"
#include "scene.h"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
#endif //ERRORCHECK
}

__host__ __device__ thrust::default_random_engine random_engine(
        int iter, int index = 0, int depth = 0) {
    //return thrust::default_random_engine(utilhash((index + 1) * iter) ^ utilhash(depth));
    //return thrust::default_random_engine(utilhash(index ^ iter ^ depth));
    //return thrust::default_random_engine(utilhash(index + iter + depth));
//    return thrust::default_random_engine(utilhash(index) ^ utilhash(iter) ^ utilhash(depth));
    int h = utilhash((1 << 31) | ((depth + 5) << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

/* Static variables for device memory, scene/camera info, etc */
static Scene *hst_scene = NULL;
static glm::vec3 *dev_image = NULL;
static Geom *dev_geom = NULL;
static Material *dev_mats = NULL;

static Pixel *dev_pixels = NULL;

/* Initialize static variables. */
void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_geom, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geom,  scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_mats, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_mats,  scene->materials.data(), scene->materials.size() *
            sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_pixels, pixelcount * sizeof(Pixel));
    hipMemset(dev_pixels, 0, pixelcount * sizeof(Pixel));

    checkCUDAError("pathtraceInit");
}

/* Clean up static variables. */
void pathtraceFree() {
    // no-ops if pointers are null
    hipFree(dev_image);
    hipFree(dev_geom);
    hipFree(dev_mats);
    hipFree(dev_pixels);

    checkCUDAError("pathtraceFree");
}

__device__ void setDOF(Ray &ray, Camera cam,
        thrust::default_random_engine rng) {
    if (cam.dof.x < 0) { return; }

    float focalLength = cam.dof.x;
    float aperture = cam.dof.y;

    thrust::uniform_real_distribution<float> u0a(0, aperture);

    glm::vec3 focusedPoint = ray.origin + ray.direction * focalLength;
    float x_offset = u0a(rng) - aperture/2;
    float y_offset = u0a(rng) - aperture/2;
    float z_offset = u0a(rng) - aperture/2;
    ray.origin += glm::vec3(x_offset, y_offset, z_offset);
    ray.direction = glm::normalize(focusedPoint - ray.origin);
}

__global__ void generateCameraRays(Camera cam, Pixel *pixels, int iter,
        glm::vec3 cam_right, glm::vec3 cam_up) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);

        // Jitter screen coordinates for anti-aliasing
        thrust::default_random_engine rng = random_engine(iter, index, 0);
        thrust::uniform_real_distribution<float> u01(0, 1);
        float jit_x = x + u01(rng);
        float jit_y = y + u01(rng);

        float screen_x = -1 * (((float) jit_x * 2.f / (float)cam.resolution.x) - 1.f);
        float screen_y = -1 * (((float) jit_y * 2.f / (float)cam.resolution.y) - 1.f);

        glm::vec3 img_point = (cam.position + cam.view)
            + (cam_right * screen_x) + (cam_up * screen_y);
        glm::vec3 ray_dir = glm::normalize(img_point - cam.position);

        Ray r;
        r.origin = cam.position;
        r.direction = ray_dir;

        setDOF(r, cam, rng);

        Pixel px;
        px.terminated = false;
        px.ray = r;
        px.color = glm::vec3(1, 1, 1);
        px.index = index;
        pixels[index] = px;
    }
}

__device__ float nearestIntersectionGeom(Ray r, Geom *geoms, int geomCount,
        Geom& nearest, glm::vec3 &intersection, glm::vec3 &normal, bool &outside) {
    float nearest_t = -1;
    for (int i = 0; i < geomCount; i++) {
        Geom g = geoms[i];
        float t = -1;

        switch (g.type) {
        case SPHERE:
            t = sphereIntersectionTest(g, r, intersection, normal, outside);
            break;
        case CUBE:
            t = boxIntersectionTest(g, r, intersection, normal, outside);
            break;
        }

        if (t > 0 && (t < nearest_t || nearest_t == -1)) {
            nearest = g;
            nearest_t = t;
        }
    }
    return nearest_t;
}

__global__ void intersect(Camera cam, glm::vec3 *image, Pixel *pixels,
        int livePixelCount, int depth, int iter,
        Geom *geoms, int geomCount, Material *mats) {
    int k = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (k < livePixelCount) {
        Pixel& px = pixels[k];

        glm::vec3 intersection = glm::vec3(0, 0, 0);
        glm::vec3 normal = glm::vec3(0, 0, 0);
        bool outside;
        Geom nearest;
        float t = nearestIntersectionGeom(px.ray, geoms, geomCount, nearest,
                intersection, normal, outside);

        if (t > 0) {
            Material m = mats[nearest.materialid];
            thrust::default_random_engine rng = random_engine(iter, px.index, depth);
            scatterRay(px.ray, px.color, intersection, normal, outside, m, rng);

            if (m.emittance > 0) {
                px.terminated = true;
            }
        }
    }
}

__global__ void debugCameraRays(Camera cam, glm::vec3 *image, Pixel *pixels) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);

        image[index] += pixels[index].ray.direction;
    }
}

__global__ void killNonterminatedRays(Camera cam, Pixel *pixels,
        int livePixelCount) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (x < livePixelCount) {
        Pixel &px = pixels[x];
        if (px.terminated == false) {
            px.terminated = true;
            px.color = glm::vec3(0, 0, 0);
        }
    }
}

__global__ void storePixels(Camera cam, glm::vec3 *image, Pixel *pixels,
        int raycount) {
    int k = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (k < raycount) {
        Pixel px = pixels[k];
        if (px.terminated == true) {
            image[px.index] += px.color;
        }
    }
}

void shootCameraRays(Camera cam, Pixel *pixels, int iter,
        dim3 blockSize, dim3 blocksPerGrid) {
    float fovy = glm::radians(cam.fov.y);
    float aspectRatio = (float)cam.resolution.x / (float)cam.resolution.y;
    float tanPhi = glm::tan(fovy);
    float fovx = glm::atan(tanPhi * aspectRatio);
    float tanTheta = glm::tan(fovx);

    glm::vec3 cam_right = glm::cross(cam.view, cam.up) * tanTheta;
    glm::vec3 cam_up = glm::cross(cam_right, cam.view) * tanPhi;

    generateCameraRays<<<blocksPerGrid, blockSize>>>(cam, pixels, iter, cam_right, cam_up);
    checkCUDAError("shootCameraRays");
}

struct terminator {
    __device__ bool operator()(const Pixel px) {
        return px.terminated == true;
    }
};

int reapPixels(Camera cam, Pixel *pixels, int livePixelCount) {
    //return StreamCompaction::Shared::compact(livePixelCount, pixels);
    Pixel *new_end = thrust::remove_if(thrust::device, pixels, pixels+livePixelCount, terminator());
    return (new_end - pixels);
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    const int blockSideLength = 8;
    const dim3 blockSize(blockSideLength, blockSideLength);
    const dim3 blocksPerGrid(
            (cam.resolution.x + blockSize.x - 1) / blockSize.x,
            (cam.resolution.y + blockSize.y - 1) / blockSize.y);

    ///////////////////////////////////////////////////////////////////////////

    shootCameraRays(cam, dev_pixels, iter, blockSize, blocksPerGrid);

    int dBlockSize = 128;
    int dGridSize = (pixelcount + dBlockSize - 1) / dBlockSize;
    int livePixelCount = pixelcount;

    for (int depth = 0; depth < traceDepth; depth++) {
        //printf("depth %d, %d pixels\n", depth, livePixelCount);
        intersect<<<dGridSize, dBlockSize>>>(
                cam, dev_image, dev_pixels,
                livePixelCount, depth, iter,
                dev_geom, hst_scene->geoms.size(), dev_mats);
        checkCUDAError("intersection");

        storePixels<<<dGridSize, dBlockSize>>>(cam, dev_image, dev_pixels, livePixelCount);
        livePixelCount = reapPixels(cam, dev_pixels, livePixelCount);
        if (livePixelCount == 0) { break; }
        dGridSize = (livePixelCount + dBlockSize - 1) / dBlockSize;
    }

    killNonterminatedRays<<<dGridSize, dBlockSize>>>(cam, dev_pixels, livePixelCount);
    storePixels<<<dGridSize, dBlockSize>>>(cam, dev_image, dev_pixels, livePixelCount);

    checkCUDAError("end");

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid, blockSize>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
