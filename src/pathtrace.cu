#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include "sceneStructs.h"
#include "scene.h"

#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#include <stream_compaction/efficient.h>

#define DI 1
#define DOF 0
#define SHOW_TIMING 0
#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
}

__host__ __device__ thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene *hst_scene = NULL;
static glm::vec3 *dev_image = NULL;
// TODO: static variables for device memory, scene/camera info, etc
// ...

static Camera *dev_camera = NULL;
static Geom *dev_geoms = NULL;
static int* dev_geoms_count = NULL;
static Material *dev_materials = NULL;
static RenderState *dev_state = NULL;
static RayState *dev_rays_begin = NULL;
static RayState *dev_rays_end = NULL;
static int *dev_light_indices = NULL;
static int *dev_light_count = NULL;

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    //std::vector<Geom> geoms = hst_scene->geoms;
    //std::vector<Material> materials = hst_scene->materials;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));
    // TODO: initialize the above static variables added above


    //Copy Camera
    hipMalloc((void**)&dev_camera, sizeof(Camera));
    hipMemcpy(dev_camera, &hst_scene->state.camera, sizeof(Camera), hipMemcpyHostToDevice);

    //Copy geometry
    hipMalloc((void**)&dev_geoms, hst_scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, hst_scene->geoms.data(), hst_scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);
    //Copy geometry count
    int geom_count = hst_scene->geoms.size();
    hipMalloc((void**)&dev_geoms_count, sizeof(int));
    hipMemcpy(dev_geoms_count, &geom_count, sizeof(int), hipMemcpyHostToDevice);

    //Copy material
    hipMalloc((void**)&dev_materials, hst_scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, hst_scene->materials.data(), hst_scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    //Copy state
    hipMalloc((void**)&dev_state, sizeof(RenderState));
    hipMemcpy(dev_state, &hst_scene->state, sizeof(RenderState), hipMemcpyHostToDevice);

    //Allocate memory for rays
    hipMalloc((void**)&dev_rays_begin, pixelcount * sizeof(RayState));
//    hipMalloc((void**)&dev_rays_end, sizeof(RayState));

    //Copy Light Indices
    hipMalloc((void**)&dev_light_indices, hst_scene->state.lightIndices.size() * sizeof(int));
    hipMemcpy(dev_light_indices, hst_scene->state.lightIndices.data(), hst_scene->state.lightIndices.size() * sizeof(int), hipMemcpyHostToDevice);

    //Copy Light Count
    int lightCount = hst_scene->state.lightIndices.size();
    hipMalloc((void**)&dev_light_count, sizeof(int));
    hipMemcpy(dev_light_count, &lightCount, sizeof(int), hipMemcpyHostToDevice);

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {

	hipFree(dev_image);
    // TODO: clean up the above static variables

    hipFree(dev_camera);
    hipFree(dev_geoms);
    hipFree(dev_geoms_count);
    hipFree(dev_materials);
    hipFree(dev_state);
    hipFree(dev_rays_begin);
//    hipFree(dev_rays_end);
    hipFree(dev_light_indices);
    hipFree(dev_light_count);

    checkCUDAError("pathtraceFree");
}

//Kernel function that gets all the ray directions
__global__ void kernGetRayDirections(Camera * camera, RayState* rays, int iter)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < camera->resolution.x && y < camera->resolution.y)
	{
		int index = x + (y * camera->resolution.x);

		//TODO : Tweak the random variable here if the image looks fuzzy
		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
		thrust::uniform_real_distribution<float> u01(0, 0.005);

		//Find the ray direction
		float sx = float(x) / ((float) (camera->resolution.x) - 1.0f);
		float sy = float(y) / ((float) (camera->resolution.y) - 1.0f);

		glm::vec3 rayDir = (camera->M - (2.0f*sx - 1.0f + u01(rng)) * camera->H - (2.0f*sy - 1.0f + u01(rng)) * camera->V);
//		glm::vec3 rayDir = (camera->M - (2.0f*sx - 1.0f) * camera->H - (2.0f*sy - 1.0f) * camera->V);

		rayDir -= camera->position;
		rayDir = glm::normalize(rayDir);

		rays[index].ray.direction = rayDir;
		rays[index].ray.origin = camera->position;
		rays[index].isAlive = true;
		rays[index].rayColor = glm::vec3(1);
		rays[index].pixelIndex = index;

//		printf("%d %d : %f %f %f\n", x, y, rayDir.x, rayDir.y, rayDir.z);
	}
}

//Kernel function that generates the Depth of field jitter
__global__ void kernJitterDOF(Camera * camera, RayState* rays, int iter)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < camera->resolution.x && y < camera->resolution.y)
	{
		int index = x + (y * camera->resolution.x);

		Ray &r = rays[index].ray;

//		bool outside;
		glm::vec3 intersectionPoint, normal;

		sphereIntersectionTest(camera->camSphere, r, intersectionPoint, normal);//, outside);

		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
		thrust::uniform_real_distribution<float> u01(-0.5, 0.5);

		r.origin += glm::vec3(camera->aperture * u01(rng), camera->aperture * u01(rng), 0);
		r.direction = glm::normalize(intersectionPoint - r.origin);
	}
}


//Kernel function that performs one iteration of tracing the path.
__global__ void kernTracePath(Camera * camera, RayState *ray, Geom * geoms, int *geomCount, int* lightIndices, int *lightCount, Material* materials, glm::vec3* image, int iter, int currDepth, int rayCount)
{
	 int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	 if(index < rayCount)
	 {
		 if(ray[index].isAlive)
		 {
			 glm::vec3 intersectionPoint = glm::vec3(0), normal = glm::vec3(0);
			 float min_t = FLT_MAX, t;
			 RayState &r = ray[index];
			 int nearestIndex = -1;
			 glm::vec3 nearestIntersectionPoint = glm::vec3(0), nearestNormal = glm::vec3(0);
//			 bool outside = false;

			 //Find geometry intersection
			 for(int i=0; i<(*geomCount); ++i)
			 {
				 if(geoms[i].type == CUBE)
				 {
					 t = boxIntersectionTest(geoms[i], r.ray, intersectionPoint, normal);//, outside);
				 }

				 else if(geoms[i].type == SPHERE)
				 {
					 t = sphereIntersectionTest(geoms[i], r.ray, intersectionPoint, normal);//, outside);
				 }

				 if(t < min_t && t > 0)//&& !outside)
				 {
					 min_t = t;
					 nearestIntersectionPoint = intersectionPoint;
					 nearestIndex = i;
					 nearestNormal = normal;
				 }
			 }

			 //If the nearest index remains unchanged, means no intersection and we can kill the ray.
			 if(nearestIndex == -1)
			 {
				 r.isAlive = false;
			 }

			 //else find the material color
			 else
			 {
				 if(materials[geoms[nearestIndex].materialid].emittance >= 1)
				 {
					 //Light source, end ray here
					 r.isAlive = false;
					 image[r.pixelIndex] += (r.rayColor
							 * materials[geoms[nearestIndex].materialid].emittance
							 * materials[geoms[nearestIndex].materialid].color);
				 }

				 else
				 {
					 thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, currDepth);

					 scatterRay(camera->position,
								 r,
								 nearestIntersectionPoint,
								 nearestNormal,
								 materials[geoms[nearestIndex].materialid],
								 rng,
								 geoms,
								 nearestIndex,
								 lightIndices,
								 lightCount);
				 }
			 }
		 }
	 }
}

__global__ void kernDirectLightPath(Camera * camera, RayState *ray, Geom * geoms, int * lightIndices, int* lightCount, Material* materials, glm::vec3* image, int iter, int currDepth, int rayCount)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(index < rayCount)
	{
		if(ray[index].isAlive)
		{
			glm::vec3 intersectionPoint, normal;
			float t;

			RayState &r = ray[index];
			int i;
			//bool outside;
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, currDepth);

			glm::vec3 pointOnLight = getRandomPointOnLight(geoms, lightIndices, lightCount, rng, i);

			r.ray.direction = glm::normalize(pointOnLight - r.ray.origin);
			t = sphereIntersectionTest(geoms[i], r.ray, intersectionPoint, normal);

			if(t > 0)
			{
				//Intersection with light, write the color
				image[r.pixelIndex] += (r.rayColor
											 * materials[geoms[i].materialid].emittance
											 * materials[geoms[i].materialid].color);
			}
		}
	}
}

struct isDead
{
	__host__ __device__ bool  operator()(const RayState r)
	{
		return (!r.isAlive);
	}
};


/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */

void pathtrace(uchar4 *pbo, int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    const int blockSideLength = 8;
    dim3 blockSize(blockSideLength, blockSideLength);
    dim3 blocksPerGrid(
            (cam.resolution.x + blockSize.x - 1) / blockSize.x,
            (cam.resolution.y + blockSize.y - 1) / blockSize.y);

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    // * For each depth:
    //   * Compute one new (ray, color) pair along each path (using scatterRay).
    //     Note that many rays will terminate by hitting a light or hitting
    //     nothing at all. You'll have to decide how to represent your path rays
    //     and how you'll mark terminated rays.
    //   * Add all of the terminated rays' results into the appropriate pixels.
    //   * Stream compact away all of the terminated paths.
    //     You may use your implementation or `thrust::remove_if` or its
    //     cousins.
    // * Finally, handle all of the paths that still haven't terminated.
    //   (Easy way is to make them black or background-colored.)

    // TODO: perform one iteration of path tracing

    //Setup initial rays
    kernGetRayDirections<<<blocksPerGrid, blockSize>>>(dev_camera, dev_rays_begin, iter);

    //Jitter rays as per Depth of field
    if(DOF)
    {
    	kernJitterDOF<<<blocksPerGrid, blockSize>>>(dev_camera, dev_rays_begin, iter);
    }

    dev_rays_end = dev_rays_begin + pixelcount;
    int rayCount = pixelcount;
    int numBlocks, numThreads = 64;

    numBlocks = (rayCount + numThreads - 1) / numThreads;

    for(int i=0; (i<traceDepth && rayCount > 0); ++i)
    {
//    	hipEvent_t start, stop;
//    	hipEventCreate(&start);
//    	hipEventCreate(&stop);
//    	hipEventRecord(start);

    	//Take one step, should make dead rays as false
    	kernTracePath<<<numBlocks, numThreads>>>(dev_camera, dev_rays_begin, dev_geoms, dev_geoms_count, dev_light_indices, dev_light_count, dev_materials, dev_image, iter, i, rayCount);

    	//Stream compaction using work efficient
//    	rayCount = StreamCompaction::Efficient::compact(rayCount, dev_rays_begin);

//    	Compact rays, dev_rays_end points to the new end
    	dev_rays_end = thrust::remove_if(thrust::device, dev_rays_begin, dev_rays_end, isDead());
    	rayCount = dev_rays_end - dev_rays_begin;

    	numBlocks = (rayCount + numThreads - 1) / numThreads;

//    	hipEventRecord(stop);
//    	hipEventSynchronize(stop);
//    	float milliseconds = 0;
//    	hipEventElapsedTime(&milliseconds, start, stop);
//    	if(SHOW_TIMING)
//    		std::cout<</*"Iter : "<<iter<<" Depth : "<<i<<" Total time in milliseconds : "<<*/milliseconds<<std::endl;
    }

    //Direct Illumination
    if(DI && rayCount > 0)
    {
    	kernDirectLightPath<<<numBlocks, numThreads>>>(dev_camera, dev_rays_begin, dev_geoms, dev_light_indices, dev_light_count, dev_materials, dev_image, iter, traceDepth, rayCount);
    }

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid, blockSize>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
