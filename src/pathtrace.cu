#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
}

__host__ __device__ thrust::default_random_engine random_engine(
        int iter, int index = 0, int depth = 0) {
    return thrust::default_random_engine(utilhash((index + 1) * iter) ^ utilhash(depth));
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene *hst_scene;
static glm::vec3 *dev_image;
static Ray* dev_rays;
static Geom* dev_geoms;
static Material* dev_materials;
// TODO: static variables for device memory, scene/camera info, etc
// ...

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));
    // TODO: initialize the above static variables added above

	const Geom* geoms = &(hst_scene->geoms)[0];
	const Material* materials = &(hst_scene->materials)[0];

	const int numObjects = hst_scene->geoms.size();
	hipMalloc((void**)&dev_rays, pixelcount*sizeof(Ray));
	hipMalloc((void**)&dev_geoms, numObjects*sizeof(Geom));
	hipMalloc((void**)&dev_materials, numObjects*sizeof(Material));

	hipMemcpy(dev_geoms, geoms, numObjects*sizeof(Geom), hipMemcpyHostToDevice);
	hipMemcpy(dev_materials, materials, numObjects*sizeof(Material), hipMemcpyHostToDevice);

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);
    // TODO: clean up the above static variables
	hipFree(dev_rays);
	hipFree(dev_geoms);
	hipFree(dev_materials);
    checkCUDAError("pathtraceFree");

}

__global__ void initRays(int iter, Camera cam, Ray* rays){
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y){
		int index = x + (y * cam.resolution.x);
		glm::vec3 left = glm::cross(cam.up, cam.view);

		thrust::default_random_engine rng = random_engine(iter, index, 0);
		thrust::uniform_real_distribution<float> u01(-0.5, 0.5);

		float res2x = cam.resolution.x / 2.0;
		float res2y = cam.resolution.y / 2.0;

		float magx = -(res2x - x + u01(rng))*sin(cam.fov.x) / res2x;
		float magy = (res2y - y + u01(rng))*sin(cam.fov.y) / res2y;

		glm::vec3 direction = cam.view + magx*left + magy*cam.up;

		rays[index].origin = cam.position;
		rays[index].direction = direction;
	}
}

__global__ void intersect(Camera cam, Ray* rays, glm::vec3* image, int numObjects, const Geom* geoms, const Material* materials){
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y){
		int index = x + (y * cam.resolution.x);

		Ray ray = rays[index];
		Ray new_ray;

		glm::vec3 normal;
		glm::vec3 intersectionPoint;
		float isIntersection;

		glm::vec3 minNormal;
		glm::vec3 minIntersectionPoint;
		float minDist = INFINITY;

		for (int i = 0; i < numObjects; i++){
			if (geoms[i].type == SPHERE){
				isIntersection = sphereIntersectionTest(geoms[i], ray, intersectionPoint, normal);
			}
			else {
				isIntersection = boxIntersectionTest(geoms[i], ray, intersectionPoint, normal);
			}

			if (isIntersection > 0 && minDist > glm::distance(ray.origin, intersectionPoint)){
				minNormal = normal;
				minIntersectionPoint = intersectionPoint;
				minDist = glm::distance(ray.origin, intersectionPoint);
			}
		}

		image[index] = minDist == INFINITY ? glm::vec3(1.0,1.0,1.0) : glm::vec3(1.0,255.0,1.0);
	}
}

/**
 * Example function to generate static and test the CUDA-GL interop.
 * Delete this once you're done looking at it!
 */
__global__ void generateStaticDeleteMe(Camera cam, int iter, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);

        thrust::default_random_engine rng = random_engine(iter, index, 0);
        thrust::uniform_real_distribution<float> u01(0, 1);

        // CHECKITOUT: Note that on every iteration, noise gets added onto
        // the image (not replaced). As a result, the image smooths out over
        // time, since the output image is the contents of this array divided
        // by the number of iterations.
        //
        // Your renderer will do the same thing, and, over time, it will become
        // smoother.
        image[index] += glm::vec3(u01(rng));
    }
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
	const Camera &cam = hst_scene->state.camera;
	const int numObjects = hst_scene->geoms.size();
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    const int blockSideLength = 8;
    const dim3 blockSize(blockSideLength, blockSideLength);
    const dim3 blocksPerGrid(
            (cam.resolution.x + blockSize.x - 1) / blockSize.x,
            (cam.resolution.y + blockSize.y - 1) / blockSize.y);

    ///////////////////////////////////////////////////////////////////////////
    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    // * For each depth:
    //   * Compute one ray along each path - many will terminate.
    //     You'll have to decide how to represent your path rays and how
    //     you'll mark terminated rays.
    //   * Add all of the terminated rays' results into the appropriate pixels.
    //   * Stream compact away all of the terminated paths.
    //     You may use your implementation or `thrust::remove_if` or its
    //     cousins.
    // * Finally, handle all of the paths that still haven't terminated.
    //   (Easy way is to make them black or background-colored.)

    // TODO: perform one iteration of path tracing
	//Ray* rays = (Ray*)malloc(pixelcount*sizeof(Ray));

	initRays<<<blocksPerGrid, blockSize>>>(iter, cam, dev_rays);
	//hipDeviceSynchronize();

	intersect<<<blocksPerGrid, blockSize>>>(cam, dev_rays, dev_image, numObjects, dev_geoms, dev_materials);
	//hipDeviceSynchronize();


    generateStaticDeleteMe<<<blocksPerGrid, blockSize>>>(cam, iter, dev_image);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid, blockSize>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
