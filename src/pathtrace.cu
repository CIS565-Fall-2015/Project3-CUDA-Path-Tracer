#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <glm/gtc/matrix_transform.hpp>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)

struct is_terminated
{
 __host__ __device__
 bool operator() (const Ray r)
 {
	return r.terminated;
 }
};


void checkCUDAErrorFn(const char *msg, const char *file, int line) {
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene *hst_scene = NULL;
static glm::vec3 *dev_image = NULL;
static Geom *dev_geoms = NULL;
static Material *dev_mats = NULL;
static Ray *dev_rayArray = NULL;

// TODO: static variables for device memory, scene/camera info, etc
// ...

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;
	const Geom *geoms = &(hst_scene->geoms)[0];
	const Material *mats = &(hst_scene->materials)[0];

	hipMalloc(&dev_geoms, pixelcount * sizeof(Geom));
	hipMalloc(&dev_mats, pixelcount * sizeof(Material));
	hipMalloc(&dev_rayArray, pixelcount * sizeof(Ray));
    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_rayArray, 0, pixelcount * sizeof(Ray));

	hipMemcpy(dev_geoms, geoms, hst_scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);
	hipMemcpy(dev_mats, mats, hst_scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);


    // TODO: initialize the above static variables added above

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
    // TODO: clean up the above static variables
	hipFree(dev_geoms);
	hipFree(dev_mats);
	hipFree(dev_rayArray);
    checkCUDAError("pathtraceFree");
}

/**
 * Example function to generate static and test the CUDA-GL interop.
 * Delete this once you're done looking at it!
 */
__global__ void generateNoiseDeleteMe(Camera cam, int iter, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);

        thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
        thrust::uniform_real_distribution<float> u01(0, 1);

        // CHECKITOUT: Note that on every iteration, noise gets added onto
        // the image (not replaced). As a result, the image smooths out over
        // time, since the output image is the contents of this array divided
        // by the number of iterations.
        //
        // Your renderer will do the same thing, and, over time, it will become
        // smoother.
        image[index] += glm::vec3(u01(rng));
    }
}

//Create ray to be shot at a pixel in the image
__global__ void kernRayGenerate(Camera cam, Ray *ray, int iter, bool dof){
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y*cam.resolution.x);
	thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
    thrust::uniform_real_distribution<float> unitDistrib(-.5f, .5f);
	thrust::uniform_real_distribution<float> dofDistrib(-1.0f, 1.0f);
	//Calculate camera's world position
	if (x < cam.resolution.x && y < cam.resolution.y) {
		glm::vec3 A = glm::cross(cam.view, cam.up);
		glm::vec3 B = glm::cross(A, cam.view);
		glm::vec3 M = cam.position + cam.view;
		float lenC = glm::length(cam.view);
		float lenA = glm::length(A);
		float lenB = glm::length(B);
		float tantheta = (float)cam.resolution.x;
		tantheta /= (float)cam.resolution.y;
		tantheta *= tan((float)glm::radians(cam.fov[1]));
	
		glm::vec3 H = (A*lenC*tantheta) / lenA;
		glm::vec3 V = (B*lenC*tan((float)glm::radians(cam.fov[1]))) / lenB;

		//Create ray with direction and origin
		//Jitter rays with uniform distribution
		//printf("%f ", unitDistrib(rng));
		float sx = ((float)x + unitDistrib(rng)) / ((float)cam.resolution.x - 1.0f);
		float sy = ((float)y + unitDistrib(rng)) / ((float)cam.resolution.y - 1.0f);
		//Get world coordinates of pixel
		glm::vec3 WC = M - (2.0f*sx - 1.0f)*H - (2.0f*sy - 1.0f)*V;
		//Get direction of ray
		glm::vec3 dir = glm::normalize(WC - cam.position);

		ray[index].origin = cam.position;
		ray[index].direction = dir;
		ray[index].color = glm::vec3(1.0, 1.0, 1.0);
		ray[index].index = index;
		ray[index].terminated = false;

		if (dof == true) {
			glm::vec3 apOff = glm::vec3(dofDistrib(rng), dofDistrib(rng), 0.0f);
			glm::vec3 new_E = cam.position + apOff;
			float focal = 11.587f; //glm::length(glm::vec3(-2.0f, 5.0f,2.0f) - new_E);
			dir *= focal;
			dir -= apOff;
			dir = glm::normalize(dir);
			ray[index].origin = new_E;
			ray[index].direction = dir;
		}
	}
}

//Helper function to get random point on cubic light
__device__ glm::vec3 getRandomPointOnCube(Geom node, int iter, int index) {
	// TODO: get the dimensions of the transformed cube in world space
	glm::vec3 dim(0.0f, 0.0f, 0.0f);
	dim = node.scale;

	// Get surface area of the cube
	float side1 = dim[0] * dim[1];		// x-y
	float side2 = dim[1] * dim[2];		// y-z
	float side3 = dim[0] * dim[2];		// x-z
	float totalArea = 2.0f * (side1 + side2 + side3);	

	thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
    thrust::uniform_real_distribution<float> unitDistrib(-.5f, .5f);
	thrust::uniform_real_distribution<float> dofDistrib(0.0f, 1.0f);

	// pick random face weighted by surface area
	float r = floor(dofDistrib(rng));
	// pick 2 random components for the point in the range (-0.5, 0.5)
	float c1 = unitDistrib(rng);
	float c2 = unitDistrib(rng);

	glm::vec3 point;
	if (r < side1 / totalArea) {				
		// x-y front
		point = glm::vec3(c1, c2, 0.5f);
	} else if (r < (side1 * 2) / totalArea) {
		// x-y back
		point = glm::vec3(c1, c2, -0.5f);
	} else if (r < (side1 * 2 + side2) / totalArea) {
		// y-z front
		point = glm::vec3(0.5f, c1, c2);
	} else if (r < (side1 * 2 + side2 * 2) / totalArea) {
		// y-z back
		point = glm::vec3(-0.5f, c1, c2);
	} else if (r < (side1 * 2 + side2 * 2 + side3) / totalArea) {
		// x-z front 
		point = glm::vec3(c1, 0.5f, c2);
	} else {
		// x-z back
		point = glm::vec3(c1, -0.5f, c2);
	}

	// TODO: transform point to world space
	glm::mat4 T(1.0f);
	T = glm::translate(T, node.translation);
				
	if (node.rotation[0] != 0){
		T = glm::rotate(T, node.rotation[0]*(PI/180.0f), glm::vec3(1,0,0));
	}
	if (node.rotation[1] != 0){
		T = glm::rotate(T, node.rotation[1]*(PI/180.0f), glm::vec3(0,1,0));
	}
	if (node.rotation[2] != 0){
		T = glm::rotate(T, node.rotation[2]*(PI/180.0f), glm::vec3(0,0,1));
	}
				
	//T = glm::scale(T, node.scale);
	glm::vec4 newPoint = T*glm::vec4(point, 1.0f);
	point = glm::vec3(newPoint[0], newPoint[1], newPoint[2]);
	return point;
}

//Helper function to get random point on spherical light
/*__device__ glm::vec3 getRandomPointOnSphere(Geom node, int iter, int index) {
	// generate u, v, in the range (0, 1)
	float u = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
	float v = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);

	float theta = 2.0f * PI * u;
	float phi = acos(2.0f * v - 1.0f);

	// find x, y, z coordinates assuming unit sphere in object space
	glm::vec3 point;
	point[0] = sin(phi) * cos(theta);
	point[1] = sin(phi) * sin(theta);
	point[2] = cos(phi);

	// TODO: transform point to world space
	glm::mat4 T(1.0f);
	T = glm::translate(T, node.translation);
				
	if (node.rotation[0] != 0){
		T = glm::rotate(T, node.rotation[0]*(PI/180.0f), glm::vec3(1,0,0));
	}
	if (node.rotation[1] != 0){
		T = glm::rotate(T, node.rotation[1]*(PI/180.0f), glm::vec3(0,1,0));
	}
	if (node.rotation[2] != 0){
		T = glm::rotate(T, node.rotation[2]*(PI/180.0f), glm::vec3(0,0,1));
	}
				
	glm::vec4 newPoint = T*glm::vec4(point, 1.0f);
	point = glm::vec3(newPoint[0], newPoint[1], newPoint[2]);
	return point;
}*/
//Helper function to find closest intersection
__device__ float closestIntersection(Ray ray, const Geom* geoms, glm::vec3 &intersectionPoint, glm::vec3 &normal, bool &outside, int &objIndex, const int numGeoms){
	glm::vec3 interPoint;
	glm::vec3 norm;
	bool out;
	float t = -1;
	float dist;
	for (int i = 0; i < numGeoms; i++) {	
		if (geoms[i].type == CUBE) {
			dist = boxIntersectionTest(geoms[i], ray, interPoint, norm, out);
		}
		else if (geoms[i].type == SPHERE) {
			dist = sphereIntersectionTest(geoms[i], ray, interPoint, norm, out);
		}
		if ((dist != -1 && dist < t) || t == -1) {
			t = dist;
			intersectionPoint = interPoint;
			normal = norm;
			outside = out;
			objIndex = i;
		
		}
	}
	return t;
		
}

//Function to find next ray
__global__ void kernPathTracer(Camera cam, Ray* rayArray, const Geom* geoms, const Material* mats, const int numGeoms, const int numMats, glm::vec3* dev_image, int iter, int depth, int traceDepth){
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * cam.resolution.x);
	//find closest intersection
	if (x < cam.resolution.x && y < cam.resolution.y && rayArray[index].terminated == false) {
		
		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, depth);
		glm::vec3 interPoint;
		glm::vec3 norm;
		bool out;
		int objIndex;
		if (depth == traceDepth) {
			dev_image[rayArray[index].index] == glm::vec3(0.0f, 0.0f, 0.0f);
			for (int i = 0; i < numGeoms; i++) {
				if (mats[geoms[i].materialid].emittance > 0 && mats[geoms[rayArray[index].geomid].materialid].emittance == 0 && mats[geoms[rayArray[index].geomid].materialid].hasReflective == 0 && mats[geoms[rayArray[index].geomid].materialid].hasRefractive == 0) {
					glm::vec3 new_pt = getRandomPointOnCube(geoms[i], iter, index);
					rayArray[index].direction = rayArray[index].origin + glm::normalize(new_pt - rayArray[index].origin);
					float t = closestIntersection(rayArray[index], geoms, interPoint, norm, out, objIndex, numGeoms);
					if (objIndex == i) {
						printf("hit light in direct");
						rayArray[index].color *= mats[geoms[i].materialid].emittance*mats[geoms[objIndex].materialid].color;
						dev_image[index] += rayArray[index].color;
					}
				}
			}
			
		}
		float t = closestIntersection(rayArray[index], geoms, interPoint, norm, out, objIndex, numGeoms);
		rayArray[index].geomid = objIndex;
		//get direction of next ray and compute new color
		if (t >= 0.0f) {
			if (mats[geoms[objIndex].materialid].emittance >= 1) {
				rayArray[index].color *= mats[geoms[objIndex].materialid].emittance*mats[geoms[objIndex].materialid].color;
				dev_image[rayArray[index].index] += rayArray[index].color;
				rayArray[index].terminated = true;
			}
			else {
				scatterRay(rayArray[index], rayArray[index].color, interPoint, norm, mats[geoms[objIndex].materialid], rng);
			}
		}
		else {
			//dev_image[index] *= glm::vec3(0.0f, 0.0f, 0.0f); //rayArray[index].color; 
			rayArray[index].terminated = true;
		}
	}	
}


/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;
	
	int numGeoms = hst_scene->geoms.size();
	int numMats = hst_scene->materials.size();
	Ray *rayArray = new Ray[pixelcount];

    const int blockSideLength = 8;
    const dim3 blockSize(blockSideLength, blockSideLength);
    const dim3 blocksPerGrid(
            (cam.resolution.x + blockSize.x - 1) / blockSize.x,
            (cam.resolution.y + blockSize.y - 1) / blockSize.y);

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray is a (ray, color) pair, where color starts as the
    //     multiplicative identity, white = (1, 1, 1).
    //   * For debugging, you can output your ray directions as colors.
    // * For each depth:
    //   * Compute one new (ray, color) pair along each path (using scatterRay).
    //     Note that many rays will terminate by hitting a light or hitting
    //     nothing at all. You'll have to decide how to represent your path rays
    //     and how you'll mark terminated rays.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //       surface.
    //     * You can debug your ray-scene intersections by displaying various
    //       values as colors, e.g., the first surface normal, the first bounced
    //       ray direction, the first unlit material color, etc.
    //   * Add all of the terminated rays' results into the appropriate pixels.
    //   * Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    // * Finally, handle all of the paths that still haven't terminated.
    //   (Easy way is to make them black or background-colored.)

    
    // TODO: perform one iteration of path tracing
	bool dof = false;
	kernRayGenerate<<<blocksPerGrid, blockSize>>>(cam, dev_rayArray, iter, dof);

	for (int i = 0; i < traceDepth + 1; i++) {
		kernPathTracer<<<blocksPerGrid, blockSize>>>(cam, dev_rayArray, dev_geoms, dev_mats, numGeoms, numMats, dev_image, iter, i, traceDepth);

		//thrust::remove_if(thrust::host, dev_rayArray, dev_rayArray + pixelcount, is_terminated());
	}
	
	hipMemcpy(rayArray, dev_rayArray, pixelcount*sizeof(Ray), hipMemcpyDeviceToHost);
	
    //generateNoiseDeleteMe<<<blocksPerGrid, blockSize>>>(cam, iter, dev_image);
	
    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid, blockSize>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
