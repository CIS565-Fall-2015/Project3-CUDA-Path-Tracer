#include <cstdio>
#include "cpu.h"
#include <ctime>
#include <hip/hip_runtime.h>

namespace StreamCompaction {
namespace CPU {

/**
 * CPU scan (prefix sum).
 */
void scan(int n, int *odata, const int *idata) {

	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	odata[0] = 0;
	for (int i=1; i<n; i++){
		odata[i] = odata[i-1] + idata[i-1];
	}

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float ms;
	hipEventElapsedTime(&ms, start, stop);
	printf("cpu scan (s) cuda: %f\n", ms/1000.0);
}

/**
 * CPU stream compaction without using the scan function.
 *
 * @returns the number of elements remaining after compaction.
 */
int compactWithoutScan(int n, int *odata, const int *idata) {
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	int c = 0;
	for (int i=0; i<n; i++){
		if(idata[i] != 0){
			odata[c] = idata[i];
			c++;
		}
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float ms;
	hipEventElapsedTime(&ms, start, stop);
	printf("cpu compact w/o scan (s) cuda: %f\n", ms/1000.0);

    return c;
}

/**
 * CPU stream compaction using scan and scatter, like the parallel version.
 *
 * @returns the number of elements remaining after compaction.
 */
int compactWithScan(int n, int *odata, const int *idata) {

	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	// Temp
	int* temp = new int[n];
	for (int i=0; i<n; i++){
		if(idata[i] != 0){
			temp[i] = 1;
		} else {
			temp[i] = 0;
		}
	}

	// Scan
	int* scan_arr = new int[n];
	scan(n, scan_arr, temp);

	// Number of elements in the final array
	int c = scan_arr[n-1] + temp[n-1];

	// Scatter
	for(int i=0; i<n; i++){
		if (temp[i] == 1){
			int oind = scan_arr[i];
			odata[oind] = idata[i];
		}
	}

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float ms;
	hipEventElapsedTime(&ms, start, stop);
	printf("cpu compact w/ scan (s) cuda: %f\n", ms/1000.0);

    return c;
}

}
}
