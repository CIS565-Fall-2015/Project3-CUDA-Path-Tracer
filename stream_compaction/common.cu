#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"



//void checkCUDAErrorFn(const char *msg, const char *file, int line) {
//    hipError_t err = hipGetLastError();
//    if (hipSuccess == err) {
//        return;
//    }
//
//    fprintf(stderr, "CUDA error");
//    if (file) {
//        fprintf(stderr, " (%s:%d)", file, line);
//    }
//    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
//    exit(EXIT_FAILURE);
//}


namespace StreamCompaction {
namespace Common {

	__global__ void kernZeroArray(int n, int * data)
	{
		int k = threadIdx.x + blockDim.x * blockIdx.x;
		if(k < n)
		{
			data[k] = 0;
		}
	}



	__global__ void kernInclusive2Exclusive(int n, int * exclusive, const int * inclusive)
	{
		int k = threadIdx.x + blockDim.x * blockIdx.x;
		if( k < n)
		{
			if(k == 0)
			{
				exclusive[k] = IDENTITY;
			}
			else
			{
				exclusive[k] = inclusive[k-1];
			}
		}
	}



	/**
	* Maps an array to an array of 0s and 1s for stream compaction. Elements
	* which map to 0 will be removed, and elements which map to 1 will be kept.
	*/
	__global__ void kernMapToBoolean(int n, int *bools, const int *idata) {
		int k = threadIdx.x + blockDim.x * blockIdx.x;
		if( k < n )
		{
			bools[k] = idata[k] != 0 ? 1 : 0;
		}
	}

	/**
	* Performs scatter on an array. That is, for each element in idata,
	* if bools[idx] == 1, it copies idata[idx] to odata[indices[idx]].
	*/
	__global__ void kernScatter(int n, int *odata,
		const int *idata, const int *bools, const int *indices) {
			int k = threadIdx.x + blockDim.x * blockIdx.x;
			if( k < n )
			{
				if(bools[k] == 1)
				{
					odata[ indices[k] ] = idata[k];
				}
			}
	}

}
}
