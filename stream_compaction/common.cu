#include "hip/hip_runtime.h"
#include "common.h"

void checkCUDAErrorFn(const char *msg, const char *file, int line) {
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
}


namespace StreamCompaction {
namespace Common {

/**
 * Maps an array to an array of 0s and 1s for stream compaction. Elements
 * which map to 0 will be removed, and elements which map to 1 will be kept.
 */
__global__ void kernMapToBoolean(int n, int *bools, const int *idata) {
	int k = threadIdx.x + (blockIdx.x * blockDim.x);

	if (k < n) {
		bools[k] = !!idata[k];
	}
}

/**
* Same as above, but handles Ray arrays.
*/
__global__ void kernMapToBoolean(int n, int *bools, const Ray *idata) {
	int k = threadIdx.x + (blockIdx.x * blockDim.x);

	if (k < n) {
		bools[k] = !!idata[k].alive; // this should be giving the same result as the above one for our alive condition
	}
}

/**
 * Performs scatter on an array. That is, for each element in idata,
 * if bools[idx] == 1, it copies idata[idx] to odata[indices[idx]].
 */
__global__ void kernScatter(int n, int *odata,
        const int *idata, const int *bools, const int *indices) {
	int k = threadIdx.x + (blockIdx.x * blockDim.x);

	if (k < n) {
		if (bools[k] == 1) {
			odata[indices[k]] = idata[k];
		}
	}
}

/**
* Same as above. But supports Rays.
*/
__global__ void kernScatter(int n, Ray *odata,
	const Ray *idata, const int *bools, const int *indices) {
	int k = threadIdx.x + (blockIdx.x * blockDim.x);

	if (k < n) {
		if (bools[k] == 1) {
			odata[indices[k]] = idata[k];
		}
	}
}

}
}
