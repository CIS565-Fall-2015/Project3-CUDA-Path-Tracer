#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Efficient {

#define blockSize 1024
int *temp_scan;
int *scan_result;

__global__ void upSweep(int n, int d, int *o_data, int *i_data) {
	int index =  (blockIdx.x * blockDim.x) + threadIdx.x;	
	if (index <= n) {
		if (index % (int)pow(2.0, d+1) == 0) {
			o_data[index-1] = i_data[index - 1 - (int)pow(2.0, d)] + i_data[index - 1];
		} 
	}
}

__global__ void downSweep(int n, int d, int *o_data, int *i_data) {
	int index =  (blockIdx.x * blockDim.x) + threadIdx.x;
	int temp = 0;
	if (index <= n) {
		if (index % (int)pow(2.0, d+1) == 0) {
			temp = i_data[index - 1 - (int)pow(2.0, d)];
			o_data[index - 1 - (int)pow(2.0, d)] = i_data[index-1];
			o_data[index-1] = temp + i_data[index - 1];
		} 
	}

}

void scan(int n, int *odata, const int *idata) {
    int d = ilog2ceil(n);
	int total = (int) pow(2.0, d);

	hipMalloc((void**)&scan_result, total * sizeof(int));
	hipMalloc((void**)&temp_scan, total * sizeof(int));
	
	hipMemcpy(temp_scan, idata, total * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(scan_result, idata, total * sizeof(int), hipMemcpyHostToDevice);

	dim3 fullBlocksPerGrid((total + blockSize - 1) / blockSize);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start);
	for (int i = 0; i < d; i++) {
		upSweep<<<fullBlocksPerGrid, blockSize>>>(total, i, scan_result, temp_scan);
		temp_scan = scan_result;
	}

	
	hipMemcpy(odata, scan_result, total * sizeof(int), hipMemcpyDeviceToHost);
	odata[total-1] = 0;

	hipMemcpy(scan_result, odata, total * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(temp_scan, odata, total * sizeof(int), hipMemcpyHostToDevice);

	for (int i = d-1; i >= 0; i--) {
		downSweep<<<fullBlocksPerGrid, blockSize>>>(total, i, scan_result, temp_scan);
		temp_scan = scan_result;
	}

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("%f milliseconds for efficient \n", milliseconds);

	hipMemcpy(odata, scan_result, total * sizeof(int), hipMemcpyDeviceToHost);
	printf("odata[n-1] %d \n", odata[total-1]);
}


/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata) {
    
	int d = ilog2ceil(n);
	int total = (int) pow(2.0, d);

	int *predicate_array;
	int *hst_predicate_array;
	int *dev_idata;
	int *compact_array;

	hipMalloc((void**)&predicate_array, total * sizeof(int));
	hipMalloc((void**)&hst_predicate_array, total * sizeof(int));
	hipMalloc((void**)&dev_idata, total * sizeof(int));
	hipMalloc((void**)&compact_array, total * sizeof(int));

	hipMemcpy(dev_idata, idata, total * sizeof(int), hipMemcpyHostToDevice);

	dim3 fullBlocksPerGrid((total + blockSize - 1) / blockSize);

	Common::kernMapToBoolean<<<fullBlocksPerGrid, blockSize>>>(total, predicate_array, dev_idata);
	hipMemcpy(hst_predicate_array, predicate_array, total * sizeof(int), hipMemcpyDeviceToHost);
	
	scan(total, odata, hst_predicate_array);
	int totalAfterCompaction = odata[total-1];
	hipMemcpy(odata, predicate_array, total * sizeof(int), hipMemcpyDeviceToHost);
	
    return totalAfterCompaction;
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int rayCompact(int n, Ray *odata, Ray *idata) {
    
	int d = ilog2ceil(n);
	int total = (int) pow(2.0, d);

	int *predicate_array;
	int *hst_predicate_array;
	int *dev_idata;
	int *compact_array;

	hipMalloc((void**)&predicate_array, total * sizeof(int));
	hipMalloc((void**)&hst_predicate_array, total * sizeof(int));
	hipMalloc((void**)&dev_idata, total * sizeof(int));
	hipMalloc((void**)&compact_array, total * sizeof(int));

	hipMemcpy(dev_idata, idata, total * sizeof(int), hipMemcpyHostToDevice);

	dim3 fullBlocksPerGrid((total + blockSize - 1) / blockSize);

	Common::kernMapRayToBoolean<<<fullBlocksPerGrid, blockSize>>>(total, predicate_array, dev_idata);
	hipMemcpy(hst_predicate_array, predicate_array, total * sizeof(int), hipMemcpyDeviceToHost);
	
	scan(total, odata, hst_predicate_array);
	int totalAfterCompaction = odata[total-1];
	hipMemcpy(odata, predicate_array, total * sizeof(int), hipMemcpyDeviceToHost);
	
    return totalAfterCompaction;
}

}
}
