#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Efficient {

#define blockSize 1024
int *temp_scan;
int *scan_result;
Ray *rays;

__global__ void upSweep(int n, int d, int *o_data, int *i_data) {
	int index =  (blockIdx.x * blockDim.x) + threadIdx.x;	
	if (index <= n) {
		if (index % (int)pow(2.0, d+1) == 0) {
			o_data[index-1] = (int)i_data[index - 1 - (int)pow(2.0, d)] + (int)i_data[index - 1];
		} 
	}
}

__global__ void downSweep(int n, int d, int *o_data, int *i_data) {
	int index =  (blockIdx.x * blockDim.x) + threadIdx.x;
	int temp = 0;
	if (index <= n) {
		if (index % (int)pow(2.0, d+1) == 0) {
			temp = i_data[index - 1 - (int)pow(2.0, d)];
			o_data[index - 1 - (int)pow(2.0, d)] = i_data[index-1];
			o_data[index-1] = temp + i_data[index - 1];
		} 
	}

}

__global__ void rayToInt(int n, int *o_data, Ray *i_data) {
	int index =  (blockIdx.x * blockDim.x) + threadIdx.x;
	int temp = 0;
	if (index <= n) {
		o_data[index] = (int) i_data[index].isAlive;
	}
}

void scan(int n, int *odata, const int *idata) {
    int d = ilog2ceil(n);
	int total = (int) pow(2.0, d);

	hipMalloc((void**)&scan_result, total * sizeof(int));
	hipMalloc((void**)&temp_scan, total * sizeof(int));
	hipMemcpy(scan_result, idata, total * sizeof(int), hipMemcpyHostToDevice);

	dim3 fullBlocksPerGrid((total + blockSize - 1) / blockSize);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start);
	for (int i = 0; i < d; i++) {
		upSweep<<<fullBlocksPerGrid, blockSize>>>(total, i, scan_result, temp_scan);
		temp_scan = scan_result;
	}

	
	scan_result[total-1] = 0;
	hipMemcpy(odata, scan_result, total * sizeof(int), hipMemcpyDeviceToHost);
	

	hipMemcpy(scan_result, odata, total * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(temp_scan, odata, total * sizeof(int), hipMemcpyHostToDevice);

	for (int i = d-1; i >= 0; i--) {
		downSweep<<<fullBlocksPerGrid, blockSize>>>(total, i, scan_result, temp_scan);
		temp_scan = scan_result;
	}

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("%f milliseconds for efficient \n", milliseconds);

	hipMemcpy(odata, scan_result, total * sizeof(int), hipMemcpyDeviceToHost);
	printf("odata[n-1] %d \n", odata[total-1]);
}


/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, Ray *odata, const Ray *idata) {
    
	int d = ilog2ceil(n);
	int total = (int) pow(2.0, d);

	int *predicate_array;
	int *hst_predicate_array;
	Ray *dev_idata;
	Ray *compacted_rays;
	int *compact_array;

	int *hst_indices;
	int *dev_indices;

	hipMalloc((void**)&predicate_array, total * sizeof(int));
	hipMalloc((void**)&hst_predicate_array, total * sizeof(int));
	hipMalloc((void**)&dev_idata, total * sizeof(Ray));
	hipMalloc((void**)&compact_array, total * sizeof(int));
	hipMalloc((void**)&hst_indices, total * sizeof(int));
	hipMalloc((void**)&dev_indices, total * sizeof(int));

	hipMemcpy(dev_idata, idata, total * sizeof(Ray), hipMemcpyHostToDevice);

	dim3 fullBlocksPerGrid((total + blockSize - 1) / blockSize);

	Common::kernMapRayToBoolean<<<fullBlocksPerGrid, blockSize>>>(total, predicate_array, 
		dev_idata);

	hipMemcpy(hst_predicate_array, predicate_array, 
		total * sizeof(int), hipMemcpyDeviceToHost);
	
	scan(total, hst_indices, hst_predicate_array);
	hipMemcpy(dev_indices, hst_indices, total * sizeof(int), hipMemcpyHostToDevice);

	int totalAfterCompaction = hst_indices[total-1];
	
	hipMalloc((void**)&compacted_rays, totalAfterCompaction * sizeof(Ray));
	Common::kernScatter<<<fullBlocksPerGrid, blockSize>>>(total, odata,
        dev_idata, predicate_array, dev_indices);

    return totalAfterCompaction;
}


}
}
