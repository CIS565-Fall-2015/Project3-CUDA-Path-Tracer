#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "common.h"
#include "efficient.h"


// used for avoid bank conflict
#define NUM_BANKS 16  
#define LOG_NUM_BANKS 4  
#define CONFLICT_FREE_OFFSET(n) ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))



int* dev_array;

namespace StreamCompaction {
	namespace Efficient {

		__global__ void kern_up_sweep(int n, int m_power, int* x) //m_power = 2^d
		{
			int index = threadIdx.x + (blockIdx.x * blockDim.x);

			index = index *m_power * 2 -1;

			if (index > 0 && index < n ) //&& ((index + 1) % (m_power * 2) == 0)
			{
				x[index] = x[index] + x[index - m_power];
			}
		}


		__global__ void kern_down_sweep(int n, int m_power, int* x) //m_power = 2^(log2(n)-1-d)
		{
			int index = threadIdx.x + (blockIdx.x * blockDim.x);

			index = index* m_power * 2 -1;

			if (index>0 && index < n ) //&& ((index + 1) % (m_power * 2) == 0)
			{
				int tmp = x[index];
				x[index] = x[index] + x[index - m_power]; //sum
				x[index - m_power] = tmp; //swap
			}

		}

		__global__ void kern_set_value(int index,int val,int* x)
		{
			x[index] = val;
		}
		/**
		 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
		 */
		void scan(int n, int *odata, const int *idata) {
			// up sweep

			int m_power = ilog2ceil(n);
			int new_n = pow(2, m_power);

			dim3 threadsPerBlock(blockSize);
			dim3 fullBlocksPerGrid((new_n + blockSize - 1) / blockSize);

			//init the array
			hipMalloc((void**)&dev_array, new_n * sizeof(int));
			checkCUDAErrorFn("hipMalloc dev_array1 failed!");

			hipMemset(dev_array, 0, new_n*sizeof(int));
			checkCUDAErrorFn("hipMemset dev_array failed!");
			
			hipMemcpy(dev_array, idata, n*sizeof(int), hipMemcpyHostToDevice);
			checkCUDAErrorFn("hipMemcpy dev_array failed!");

			
			int* pow_2_d  =new int[m_power];
			int* pow_2_log2n_minus_d = new int[m_power];
			for (int d = 0; d < m_power; d++)
			{
				pow_2_d[d] = pow(2, d);
				
				int nn = m_power - 1 - d;
				
				pow_2_log2n_minus_d[d] = pow(2, nn);

			}
			
			
			
			//cuda event init
			hipEvent_t start, stop;
			hipEventCreate(&start);
			hipEventCreate(&stop);
			float milliseconds = 0;

			hipEventRecord(start);
			
			//up sweep
			for (int d = 0; d < m_power; d++)
			{
				//int pow_2_d = pow(2, d);
				kern_up_sweep << <fullBlocksPerGrid, threadsPerBlock >> >(new_n, pow_2_d[d], dev_array);
			}

			//down sweep
			
			kern_set_value << <1, 1 >> > (new_n - 1, 0, dev_array); //insert 0
			
			
			for (int d = 0; d < m_power; d++)
			{
				/*int nn = m_power - 1 - d;
				int pow_2_log2n_minus_d = pow(2, nn);*/

				kern_down_sweep << <fullBlocksPerGrid, threadsPerBlock >> >(new_n, pow_2_log2n_minus_d[d], dev_array);
			}

			
			hipEventRecord(stop);
			hipEventSynchronize(stop);
			milliseconds = 0;
			hipEventElapsedTime(&milliseconds, start, stop);
			std::cout << "efficient method: " << milliseconds << "ms" << std::endl;
			
			//copy data
			hipMemcpy(odata, dev_array, n*sizeof(int), hipMemcpyDeviceToHost);

		}






		/**
		 * Performs stream compaction on idata, storing the result into odata.
		 * All zeroes are discarded.
		 *
		 * @param n      The number of elements in idata.
		 * @param odata  The array into which to store elements.
		 * @param idata  The array of elements to compact.
		 * @returns      The number of elements remaining after compaction.
		 */
		int compact(int n, int *odata, const int *idata) {
			
			dim3 threadsPerBlock(blockSize);
			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

			//copy data to device
			int* dev_idata;
			hipMalloc((void**)&dev_idata, n * sizeof(int));
			checkCUDAErrorFn("hipMalloc dev_idata failed!");

			hipMemset(dev_idata, 0, n*sizeof(int));
			checkCUDAErrorFn("hipMemset dev_idata failed!");

			hipMemcpy(dev_idata, idata, n*sizeof(int), hipMemcpyHostToDevice);
			checkCUDAErrorFn("hipMemcpy dev_idata failed!");

			
			// map the idata to bools
			int* dev_bools;
			hipMalloc((void**)&dev_bools, n * sizeof(int));
			checkCUDAErrorFn("hipMalloc dev_bools failed!");

			hipMemset(dev_bools, 0, n*sizeof(int));
			checkCUDAErrorFn("hipMemset dev_bools failed!");

			StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid ,threadsPerBlock>> >(n, dev_bools, dev_idata);

			//scan the bools to get the indices
			

			int* host_bools = new int[n];
			hipMemcpy(host_bools, dev_bools, n*sizeof(int), hipMemcpyDeviceToHost);
			int* host_indices = new int[n];
			
			scan(n, host_indices, host_bools);  //input is host data

			int* dev_indices;
			hipMalloc((void**)&dev_indices, n * sizeof(int));
			checkCUDAErrorFn("hipMalloc dev_indices failed!");

			hipMemcpy(dev_indices, host_indices, n*sizeof(int), hipMemcpyHostToDevice);
			checkCUDAErrorFn("hipMemcpy dev_indices failed!");

			//run scatter
			int* dev_odata;
			hipMalloc((void**)&dev_odata, n * sizeof(int));
			checkCUDAErrorFn("hipMalloc dev_bools failed!");

			hipMemset(dev_odata, 0, n*sizeof(int));
			checkCUDAErrorFn("hipMemset dev_bools failed!");
			
			StreamCompaction::Common::kernScatter << < fullBlocksPerGrid, threadsPerBlock >> > (n, dev_odata,dev_idata,dev_bools,dev_indices);
			
			//copy back to host
			hipMemcpy(odata, dev_odata, n*sizeof(int), hipMemcpyDeviceToHost);

			return host_indices[n - 1]+host_bools[n-1]; //num of non-zero
			

			
		}


		__global__ void kern_prescan(int *g_idata, int n)
		{
			extern __shared__ int temp[];  // allocated on invocation  
			int thid = threadIdx.x;
			int offset = 1;

			if (thid < n)
			{
				temp[thid] = g_idata[thid]; // load input into shared memory  

			
				for (int d = n >> 1; d > 0; d >>= 1)                    // build sum in place up the tree  
				{
					__syncthreads();
					if (thid < d)
					{
						int ai = offset*(2 * thid + 1) - 1;
						int bi = offset*(2 * thid + 2) - 1;

						temp[bi] += temp[ai];

					}
					offset *= 2;
				}

				if (thid == 0)
				{
					temp[n - 1] = 0;
				} // clear the last element  


				for (int d = 1; d < n; d *= 2) // traverse down tree & build scan  
				{
					offset >>= 1;
					__syncthreads();
					if (thid < d)
					{
						int ai = offset*(2 * thid + 1) - 1;
						int bi = offset*(2 * thid + 2) - 1;


						int t = temp[ai];
						temp[ai] = temp[bi];
						temp[bi] += t;
					}
				}

				__syncthreads(); //make sure all threads are done with writing result

				g_idata[thid] = temp[thid]; // write results to device memory  
				
			}
		}
			
		


		void scan_share_mem(int n, int *odata, const int *idata)
		{

			int m_power = ilog2ceil(n);
			int new_n = pow(2, m_power);

			dim3 threadsPerBlock(512);
			//dim3 fullBlocksPerGrid((new_n + blockSize - 1) / blockSize);

			//init the array
			hipMalloc((void**)&dev_array, new_n * sizeof(int));
			checkCUDAErrorFn("hipMalloc dev_array1 failed!");

			hipMemset(dev_array, 0, new_n*sizeof(int));
			checkCUDAErrorFn("hipMemset dev_array failed!");

			hipMemcpy(dev_array, idata, n*sizeof(int), hipMemcpyHostToDevice);
			checkCUDAErrorFn("hipMemcpy dev_array failed!");


			//cuda event init
			hipEvent_t start, stop;
			hipEventCreate(&start);
			hipEventCreate(&stop);
			float milliseconds = 0;

			hipEventRecord(start);

			//invoke prescan
			kern_prescan << <1,threadsPerBlock, new_n * sizeof(int)>> >(dev_array , new_n);


			hipEventRecord(stop);
			hipEventSynchronize(stop);
			milliseconds = 0;
			hipEventElapsedTime(&milliseconds, start, stop);
			std::cout << "efficient method: " << milliseconds << "ms" << std::endl;

			//copy data
			hipMemcpy(odata, dev_array, n*sizeof(int), hipMemcpyDeviceToHost);

		}


		int compact_share_mem(int n, int *odata, const int *idata)
		{
			dim3 threadsPerBlock(blockSize);
			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

			//copy data to device
			int* dev_idata;
			hipMalloc((void**)&dev_idata, n * sizeof(int));
			checkCUDAErrorFn("hipMalloc dev_idata failed!");

			hipMemset(dev_idata, 0, n*sizeof(int));
			checkCUDAErrorFn("hipMemset dev_idata failed!");

			hipMemcpy(dev_idata, idata, n*sizeof(int), hipMemcpyHostToDevice);
			checkCUDAErrorFn("hipMemcpy dev_idata failed!");


			// map the idata to bools
			int* dev_bools;
			hipMalloc((void**)&dev_bools, n * sizeof(int));
			checkCUDAErrorFn("hipMalloc dev_bools failed!");

			hipMemset(dev_bools, 0, n*sizeof(int));
			checkCUDAErrorFn("hipMemset dev_bools failed!");

			StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, threadsPerBlock >> >(n, dev_bools, dev_idata);

			//scan the bools to get the indices


			int* host_bools = new int[n];
			hipMemcpy(host_bools, dev_bools, n*sizeof(int), hipMemcpyDeviceToHost);
			int* host_indices = new int[n];

			scan_share_mem(n, host_indices, host_bools);  //input is host data

			int* dev_indices;
			hipMalloc((void**)&dev_indices, n * sizeof(int));
			checkCUDAErrorFn("hipMalloc dev_indices failed!");

			hipMemcpy(dev_indices, host_indices, n*sizeof(int), hipMemcpyHostToDevice);
			checkCUDAErrorFn("hipMemcpy dev_indices failed!");

			//run scatter
			int* dev_odata;
			hipMalloc((void**)&dev_odata, n * sizeof(int));
			checkCUDAErrorFn("hipMalloc dev_bools failed!");

			hipMemset(dev_odata, 0, n*sizeof(int));
			checkCUDAErrorFn("hipMemset dev_bools failed!");

			StreamCompaction::Common::kernScatter << < fullBlocksPerGrid, threadsPerBlock >> > (n, dev_odata, dev_idata, dev_bools, dev_indices);

			//copy back to host
			hipMemcpy(odata, dev_odata, n*sizeof(int), hipMemcpyDeviceToHost);

			return host_indices[n - 1] + host_bools[n - 1]; //num of non-zero
		}


	}
}

