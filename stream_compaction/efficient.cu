#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Efficient {

__global__ void up_sweep(int n, int d, int *data) {
	int k = threadIdx.x + (blockIdx.x * blockDim.x);

	if (k < n) {
		int p2d = pow(2.0, (double)d);
		int p2da1 = pow(2.0, (double)(d + 1));

		if (k % p2da1 == 0) {
			data[k + p2da1 - 1] += data[k + p2d - 1];
		}
	}	
}

__global__ void down_sweep(int n, int d, int *data) {
	int k = threadIdx.x + (blockIdx.x * blockDim.x);

	if (k < n) {
		int p2d = pow(2.0, (double)d);
		int p2da1 = pow(2.0, (double)(d + 1));

		if (k % p2da1 == 0) {
			int temp = data[k + p2d - 1];
			data[k + p2d - 1] = data[k + p2da1 - 1];
			data[k + p2da1 - 1] += temp;
		}
	}
}

void padArrayRange(int start, int end, int *a) {
	for (int i = start; i < end; i++) {
		a[i] = 0;
	}
}
/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
float scan(int n, int *odata, const int *idata) {
	int m = pow(2, ilog2ceil(n));
	int *new_idata = (int*)malloc(m * sizeof(int));
	dim3 fullBlocksPerGrid((m + blockSize - 1) / blockSize);
	dim3 threadsPerBlock(blockSize);

	hipEvent_t start, stop;
	float ms_time = 0.0f;
	float ms_total_time = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Expand array to next power of 2 size
	for (int i = 0; i < n; i++) {
		new_idata[i] = idata[i];
	}
	padArrayRange(n, m, new_idata);

	// Can use one array for input and output in this implementation
	int *dev_data;
	hipMalloc((void**)&dev_data, m * sizeof(int));
	hipMemcpy(dev_data, new_idata, m * sizeof(int), hipMemcpyHostToDevice);

	// Execute scan on device
	hipEventRecord(start);
	for (int d = 0; d < ilog2ceil(n); d++) {
		up_sweep<<<fullBlocksPerGrid, threadsPerBlock>>>(n, d, dev_data);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&ms_time, start, stop);
	ms_total_time += ms_time;
	ms_time = 0.0f;

	hipMemset((void*)&dev_data[m - 1], 0, sizeof(int));
	hipEventRecord(start);
	for (int d = ilog2ceil(n) - 1; d >= 0; d--) {
		down_sweep<<<fullBlocksPerGrid, threadsPerBlock>>>(n, d, dev_data);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&ms_time, start, stop);
	ms_total_time += ms_time;

	hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_data);
	free(new_idata);

	return ms_total_time;
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata) {
	int *bools = (int*)malloc(n * sizeof(int));
	int *scan_data = (int*)malloc(n * sizeof(int));
	int num_remaining = -1;
	dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
	dim3 threadsPerBlock(blockSize);

	hipEvent_t start, stop;
	float ms_time = 0.0f;
	float ms_total_time = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int *dev_bools;
	int *dev_idata;
	int *dev_odata;
	int *dev_scan_data;

	hipMalloc((void**)&dev_bools, n * sizeof(int));
	hipMalloc((void**)&dev_idata, n * sizeof(int));
	hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**)&dev_odata, n * sizeof(int));
	hipMalloc((void**)&dev_scan_data, n * sizeof(int));

	// Map to boolean
	hipEventRecord(start);
	StreamCompaction::Common::kernMapToBoolean<<<fullBlocksPerGrid, threadsPerBlock>>>(n, dev_bools, dev_idata);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&ms_time, start, stop);
	ms_total_time += ms_time;
	ms_time = 0.0f;

	hipMemcpy(bools, dev_bools, n * sizeof(int), hipMemcpyDeviceToHost);

	// Execute the scan
	ms_total_time += scan(n, scan_data, bools);
	num_remaining = scan_data[n - 1] + bools[n - 1];

	// Execute the scatter
	hipMemcpy(dev_scan_data, scan_data, n * sizeof(int), hipMemcpyHostToDevice);
	hipEventRecord(start);
	StreamCompaction::Common::kernScatter<<<fullBlocksPerGrid, threadsPerBlock>>>(n, dev_odata, dev_idata, dev_bools, dev_scan_data);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&ms_time, start, stop);
	ms_total_time += ms_time;
	printf("CUDA execution time for stream compaction: %.5fms\n", ms_total_time);

	hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_bools);
	hipFree(dev_idata);
	hipFree(dev_odata);
	hipFree(dev_scan_data);
	free(bools);
	free(scan_data);

	return num_remaining;
}

}
}
