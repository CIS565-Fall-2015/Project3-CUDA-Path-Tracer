#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Efficient {

hipEvent_t start, stop;

static void setup_timer_events() {
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
}

static float teardown_timer_events() {
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return milliseconds;
}

// TODO: __global__

__global__ void upsweep_step(int d_offset_plus, int d_offset, int *x) {
	int k = threadIdx.x + (blockIdx.x * blockDim.x);
	if (k % d_offset_plus) {
		return;
	}
	x[k + d_offset_plus - 1] += x[k + d_offset - 1];
}

__global__ void downsweep_step(int d_offset_plus, int d_offset, int *x) {
	int k = threadIdx.x + (blockIdx.x * blockDim.x);
	if (k % d_offset_plus) {
		return;
	}
	int t = x[k + d_offset - 1];
	x[k + d_offset - 1] = x[k + d_offset_plus - 1];
	x[k + d_offset_plus - 1] += t;
}

__global__ void fill_by_value(int val, int *x) {
	int k = threadIdx.x + (blockIdx.x * blockDim.x);
	x[k] = val;
}

static void setup_dimms(dim3 &dimBlock, dim3 &dimGrid, int n) {
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	int tpb = deviceProp.maxThreadsPerBlock;
	int blockWidth = fmin(n, tpb);
	int blocks = 1;
	if (blockWidth != n) {
		blocks = n / tpb;
		if (n % tpb) {
			blocks ++;
		}
	}

	dimBlock = dim3(blockWidth);
	dimGrid = dim3(blocks);
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {

	// we'll need to pad the device memory with 0s to get a power of 2 array size.
	int logn = ilog2ceil(n);
	int pow2 = (int)pow(2, logn);

	dim3 dimBlock;
	dim3 dimGrid;
	setup_dimms(dimBlock, dimGrid, pow2);

	int *dev_x;
	hipMalloc((void**)&dev_x, sizeof(int) * pow2);
	fill_by_value <<<dimGrid, dimBlock >>>(0, dev_x);
	// copy everything in idata over to the GPU.
	hipMemcpy(dev_x, idata, sizeof(int) * n, hipMemcpyHostToDevice);

	// up sweep and down sweep
	up_sweep_down_sweep(pow2, dev_x, -1, -1);

	hipMemcpy(odata, dev_x, sizeof(int) * n, hipMemcpyDeviceToHost);
	hipFree(dev_x);
}

// exposed up sweep and down sweep. expects powers of two!
void up_sweep_down_sweep(int n, int *dev_data1, int blocksPerGrid, int blockSize) {
	int logn = ilog2ceil(n);

	dim3 dimBlock(blockSize, 1);
	dim3 dimGrid(blocksPerGrid, 1);
	if (blockSize < 0 && blocksPerGrid < 0)
		setup_dimms(dimBlock, dimGrid, n);

	// Up Sweep
	for (int d = 0; d < logn; d++) {
		int d_offset_plus = (int)pow(2, d + 1);
		int d_offset = (int)pow(2, d);
		upsweep_step << <dimGrid, dimBlock >> >(d_offset_plus, d_offset, dev_data1);
	}

	// Down-Sweep
	hipMemset(&dev_data1[n - 1], 0, sizeof(int) * 1);
	for (int d = logn - 1; d >= 0; d--) {
		int d_offset_plus = (int)pow(2, d + 1);
		int d_offset = (int)pow(2, d);
		downsweep_step << <dimGrid, dimBlock >> >(d_offset_plus, d_offset, dev_data1);
	}
}

__global__ void temporary_array(int *x, int *temp) {
	int k = threadIdx.x + (blockIdx.x * blockDim.x);
	temp[k] = (x[k] != 0);
}

__global__ void scatter(int *x, int *trueFalse, int* scan, int *out) {
	int k = threadIdx.x + (blockIdx.x * blockDim.x);
	if (trueFalse[k]) {
		out[scan[k]] = x[k];
	}
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata) {
	int logn = ilog2ceil(n);
	int pow2 = (int)pow(2, logn);

	dim3 dimBlock;
	dim3 dimGrid;
	setup_dimms(dimBlock, dimGrid, pow2);

	int *dev_x;
	int *dev_tmp;
	int *dev_scatter;
	int *dev_scan;

	hipMalloc((void**)&dev_x, sizeof(int) * pow2);
	hipMalloc((void**)&dev_tmp, sizeof(int) * pow2);
	hipMalloc((void**)&dev_scan, sizeof(int) * pow2);
	hipMalloc((void**)&dev_scatter, sizeof(int) * pow2);

	// 0 pad up to a power of 2 array length.
	// copy everything in idata over to the GPU.
	fill_by_value << <dimGrid, dimBlock >> >(0, dev_x);
	hipMemcpy(dev_x, idata, sizeof(int) * n, hipMemcpyHostToDevice);

    // Step 1: compute temporary true/false array
	temporary_array <<<dimGrid, dimBlock >>>(dev_x, dev_tmp);

	// Step 2: run efficient scan on the tmp array
	hipMemcpy(dev_scan, dev_tmp, sizeof(int) * pow2, hipMemcpyDeviceToDevice);
	up_sweep_down_sweep(pow2, dev_scan, -1, -1);

	// Step 3: scatter
	scatter <<<dimGrid, dimBlock >>>(dev_x, dev_tmp, dev_scan, dev_scatter);

	hipMemcpy(odata, dev_scatter, sizeof(int) * n, hipMemcpyDeviceToHost);

	int last_index;
	hipMemcpy(&last_index, dev_scan + (n - 1), sizeof(int),
		hipMemcpyDeviceToHost);

	int last_true_false;
	hipMemcpy(&last_true_false, dev_tmp + (n - 1), sizeof(int),
		hipMemcpyDeviceToHost);

	hipFree(dev_x);
	hipFree(dev_tmp);
	hipFree(dev_scan);
	hipFree(dev_scatter);

	return last_index + last_true_false;
}

__global__ void block_upsweep(int n, int *dev_data) {
	// parallel reduction with some modifications
	// in place of:
	// 0  1  2  3  4  5  6  7   stride = 1  
	// 1     5     9     13     stride = 2 
	// 6           22           stride = 4 
	// 28                              
	//
	// we want:
	// 0  1  2  3  4  5  6  7   stride = 1
	//    1     5     9     13  stride = 2
	//          6           22  stride = 4
	//                      28
	//
	// want to do stuff at indices:
	// 1  3  5  6  7 -> stride to get here is 1
	// 3 7 -> stride to get here is 2
	// 7 -> stride to get here is 4
	//
	// use if((t + 2) % (2 * stride) == 1)
	// needs to produce something more like an "upsweep" than a traditional parallel reduction

	unsigned int t = threadIdx.x; // we're indexing shared memory, so no need for +(blockIdx.x * blockDim.x);

	// load into shared memory from provided pointer
	// we know dev_data is spread over the entire grid
	// so start is blockId.x * blockDim.x, size i blockDim.x
	__shared__ int block_data[blockDim.x];
	block_data[t] = dev_data[t + blockIdx.x * blockDim.x];
	// for each stage:
	for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
		// syncthreads to make sure all threads have transferred relevant data
		__syncthreads();
		// compute partial
		if ((t + 2) % (2 * stride) == 1) {
			partialSum[t] += partialSum[t - stride];
		}
	}
	__syncthreads(); // make sure all threads are done computing
	// write the data out
	dev_data[t + blockIdx.x * blockDim.x] = block_data[t];
}

__global__ void block_downsweep() {

}

void efficient_scan(int n, int *dev_data, int blocksPerGrid, int blockSize) {
	// break up into blocks. pad with zeros so we have pow 2
	// run scan on each block (upsweep downsweep)
	// accumulate block sums into an array of sums.
	// scan block sums to compute block increments. if it's too big for one block, recurse (omg)
	// add block increments to each element in the corresponding block. stop at n, don't pile on zeros
	// return!
}

}
}
