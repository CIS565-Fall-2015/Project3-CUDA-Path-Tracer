#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "efficient.h"
#include <iostream>

namespace StreamCompaction {
namespace Efficient {

const int threadCount = 256;

void printArray(int n, int * a)
{
	printf("\n");
	for(int i=0; i<n; ++i)
		printf("%d ", a[i]);
	printf("\n");
}

__global__ void setK(int * k, int * data, int index)
{
	(*k) = data[index];
}

__global__ void blockWiseScan(int n, int *odata, int *idata)
{
	//Reference-> http://http.developer.nvidia.com/GPUGems3/gpugems3_ch39.html

	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if(index < n)
	{
		//Do block exclusive scans
		__shared__ int data[threadCount];

		unsigned int t = threadIdx.x;
		n = blockDim.x;

		data[t] = idata[index];
		int offset = 1;

		for (int d = n>>1; d > 0; d >>= 1)  // build sum in place up the tree
		{
			__syncthreads();
			if (t < d)
			{
				int ai = offset * ((t<<1)+1) - 1;
				int bi = offset * ((t<<1)+2) - 1;

				data[bi] += data[ai];
			}
			offset <<= 1;
		}

		if (t == 0) { data[n - 1] = 0; } // clear the last element

		for (int d = 1; d < n; d <<= 1) // traverse down tree & build scan
		{
		     offset >>= 1;
		     __syncthreads();

		     if (t < d)
		     {
		    	 int ai = offset * ((t<<1)+1) - 1;
		    	 int bi = offset * ((t<<1)+2) - 1;

		    	 float t = data[ai];
		    	 data[ai] = data[bi];
		    	 data[bi] += t;
		     }
		}

		odata[index] = data[t];
	}
}

__global__ void createTemp(int * odata, int *idata, int * temp, int numThreads)
{
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	temp[index] = odata[(index+1) * numThreads - 1] + idata[(index+1) * numThreads - 1];
}

__global__ void updateidata(int n, int *odata, int *temp_data)
{
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	odata[index] += temp_data[blockIdx.x];
}

void exclusiveScan(int n, int *odata, int *idata, int numBlocks, int numThreads)
{
	blockWiseScan<<<numBlocks, numThreads>>>(n, odata, idata);

	//Then we have to recurse and solve the odata array, So create a new array and solve.
	int *dev_temp,
		*dev_odata;
	int p = ilog2ceil(numBlocks);
	int	fullN = pow(2, p);

	hipMalloc((void**)&dev_temp, fullN * sizeof(int));
	hipMalloc((void**)&dev_odata, fullN * sizeof(int));

	hipMemset(dev_temp, 0, fullN * sizeof(int));
	createTemp<<<1, numBlocks>>>(odata, idata, dev_temp, numThreads);

	int newN = numBlocks;
	int newNumBlocks = (numBlocks + numThreads -1) / numThreads;

	if(numBlocks > numThreads)
	{
		exclusiveScan(newN, dev_odata, dev_temp, newNumBlocks, numThreads);
	}

	else
	{
		blockWiseScan<<<newNumBlocks, numThreads>>>(newN, dev_odata, dev_temp);
	}


	updateidata<<<numBlocks, numThreads>>>(n, odata, dev_odata);
	hipFree(dev_temp);
	hipFree(dev_odata);
}

int compact(int n, RayState *odata, RayState *idata) {

	std::cout<<n<<std::endl;
	int oriN = n;

	int p = ilog2ceil(n);
	n = pow(2, p);
	std::cout<<n<<std::endl;

	int numThreads = threadCount,
		numBlocks = (n + numThreads - 1) / numThreads;

	int	*dev_k,
		*dev_scanData,
		*dev_temp;
	int *printData = new int[n];

	hipMalloc((void**)&dev_k, sizeof(int));
	hipMalloc((void**)&dev_scanData, n * sizeof(int));
	hipMalloc((void**)&dev_temp, n * sizeof(int));


	StreamCompaction::Common::kernMapToBoolean<<<numBlocks, numThreads>>>(n, dev_scanData, idata);

	hipMemcpy(printData, dev_scanData, n * sizeof(int), hipMemcpyDeviceToHost);
	printArray(n, printData);

	exclusiveScan(n, dev_temp, dev_scanData, numBlocks, numThreads);

	hipMemcpy(printData, dev_temp, n * sizeof(int), hipMemcpyDeviceToHost);
	printArray(n, printData);

	setK<<<1,1>>>(dev_k, dev_temp, n-1);
	int *k = new int;
	hipMemcpy(k, dev_k, sizeof(int), hipMemcpyDeviceToHost);

	StreamCompaction::Common::kernScatter<<<numBlocks, numThreads>>>(n, odata, idata, dev_scanData, dev_temp);

	hipMemcpy(idata, odata, oriN * sizeof(RayState), hipMemcpyDeviceToDevice);

	std::cout<<*k<<std::endl;

	hipFree(dev_scanData);
	hipFree(dev_k);
	hipFree(dev_temp);
	return (*k);
}


}
}


namespace StreamCompaction {
namespace Common {

__global__ void kernMapToBoolean(int n, int *bools, const RayState *idata) {

	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if(index < n)
	{
//		if(idata[index].isAlive)
//		{
//			printf("Here\n");
//		}
	bools[index] = (idata[index].isAlive) ? 1 : 0;
	}
}

/**
 * Performs scatter on an array. That is, for each element in idata,
 * if bools[idx] == 1, it copies idata[idx] to odata[indices[idx]].
 */
__global__ void kernScatter(int n, RayState *odata,
        const RayState *idata, const int *bools, const int *indices) {
    // TODO

	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if(index < n)
	{
		if(bools[index] == 1)
		{
			int i = indices[index];
			odata[i].isAlive = idata[index].isAlive;
			odata[i].pixelIndex = idata[index].pixelIndex;
			odata[i].rayColor = idata[index].rayColor;
			odata[i].ray.direction = idata[index].ray.direction;
			odata[i].ray.origin = idata[index].ray.origin;
		}
	}
}

}
}
