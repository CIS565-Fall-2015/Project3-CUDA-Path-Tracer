#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
namespace Naive {

__global__ void kern_scan(int n, int d, int *idata, int *odata) {
	int k = threadIdx.x + (blockIdx.x * blockDim.x);

	if (k < n) {
		if (k >= (int)pow(2.0, (double)(d - 1))) {
			odata[k] = idata[k - (int)pow(2.0, (double)(d - 1))] + idata[k];
		}
		else {
			odata[k] = idata[k];
		}
	}
}

void padArrayRange(int start, int end, int *a) {
	for (int i = start; i < end; i++) {
		a[i] = 0;
	}
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
	int m = pow(2, ilog2ceil(n));
	int *new_idata = (int*)malloc(m * sizeof(int));
	dim3 fullBlocksPerGrid((m + blockSize - 1) / blockSize);
	dim3 threadsPerBlock(blockSize);

	hipEvent_t start, stop;
	float ms_time = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// Expand array to next power of 2 size
	for (int i = 0; i < n; i++) {
		new_idata[i] = idata[i];
	}
	padArrayRange(n, m, new_idata);

	int *dev_idata;
	int *dev_odata;

	hipMalloc((void**)&dev_idata, m * sizeof(int));
	hipMemcpy(dev_idata, new_idata, m * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**)&dev_odata, m * sizeof(int));
	
	
	// Execute scan on device
	hipEventRecord(start);
	for (int d = 1; d <= ilog2ceil(n); d++) {
		kern_scan<<<fullBlocksPerGrid, threadsPerBlock>>>(n, d, dev_idata, dev_odata);
		dev_idata = dev_odata;
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&ms_time, start, stop);
	printf("CUDA execution time for naive scan: %.5fms\n", ms_time);

	odata[0] = 0;
	hipMemcpy(odata + 1, dev_odata, (m * sizeof(int)) - sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_idata);
	hipFree(dev_odata);
	free(new_idata);
}

}
}


