#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
namespace Naive {

#define blockSize 128
int *scan_result;
int *temp_scan;
int *shifted_result;

// TODO: __global__

__global__ void prefixSum(int n, int d, int *o_data, int *i_data) {
	int index =  (blockIdx.x * blockDim.x) + threadIdx.x;
	
	if (index < n) {
		if (index >= (int)pow(2.0, d-1)) {
			o_data[index] = i_data[index - (int)pow(2.0, d-1)] + i_data[index];
		} 
	}
}



/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {

	int d = ilog2ceil(n);

	hipMalloc((void**)&scan_result, n * sizeof(int));
	hipMalloc((void**)&temp_scan, n * sizeof(int));
	hipMalloc((void**)&shifted_result, n * sizeof(int));
	
	hipMemcpy(temp_scan, idata, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(scan_result, idata, n * sizeof(int), hipMemcpyHostToDevice);

	dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start);
	for (int i = 1; i <= d; i++) {
		prefixSum<<<fullBlocksPerGrid, blockSize>>>(n, i, scan_result, temp_scan);
		temp_scan = scan_result;
	}
	hipEventRecord(stop);

	hipMemcpy(odata, scan_result, n * sizeof(int), hipMemcpyDeviceToHost);

	//shift right
	for(int i = n-1; i >= 0; i--) {
		odata[i] = odata[i-1];
	}
	odata[0] = 0;

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("%f milliseconds for naive \n", milliseconds);
	
	cleanUp();
   
}

void cleanUp() {
	hipFree(scan_result);
	hipFree(temp_scan);
}

}
}
